#include "hip/hip_runtime.h"
#include "functions.h"
#include "debugger.h"

extern PrimeHeader pheader;
extern GpuHandler gpu_data;
extern const char* PRIME_FILENAME;


using namespace std;


long find_number_of_gpus() {
    // System command to find number of GPUs attached 
    // find /proc/driver/nvidia/gpus -type d | wc -l

    char cmd[100] = "find /proc/driver/nvidia/gpus -type d | wc -l\0";
    array<char, 128> buffer;
    string result;
    shared_ptr<FILE> pipe(popen(cmd, "r"), pclose);
    if (!pipe) throw runtime_error("popen() failed!");
    while (!feof(pipe.get())) {
        if (fgets(buffer.data(), 128, pipe.get()) != nullptr)
            result += buffer.data();
    }
    long number_of_gpus = (long)stoi(result);
    number_of_gpus--; // The systems command returns a value which is
    // one more than the actual number of GPUs.
    return (number_of_gpus);

    // Function Complete --KTB
}




void start_info() {
    // Will print all the stats about the program like
    // Number of GPU being used.
    // End Number being Calculated, and so on.

    green_start();
    cout << "\n\n\n\n\n\n\n\n\n\nProgram Start\n";
    color_reset();

}



void end_info() {
    // Statistics about the program goes here


    cout << endl<< endl<< endl;
}



void iteration_info() {
    // Statistics about every iteration of the program goes here


}


void printList(uint64_cu* ilist, uint64_cu len){
    printf("\n(START, length-> %llu)\n", len);
    int c = 0 ;
    for(uint64_cu index=0; index<len ; index++){
        printf("%llu ",ilist[index]);
        c++;
        if(c==100){
            printf("\n");
            c = 0;
        }
    }
    printf("\nEND \n");
}

ThreadRetValue* kernelLauncher(int gpu_id) {
    /*
       Convention for naming variables:
len: relates to number of elements
size: relates to size of memory
     */

    cout <<endl<< "#################### START of gpu_id "<< gpu_id << " ####################"<<endl;

    uint64_cu IL_len =  gpu_data.IL_end - gpu_data.IL_start + 1;
    int total_gpus = gpu_data.gpus;
    uint64_cu PL_len = pheader.length;

    // Declare all pointers to GPU:
    int *d_IL;                                                                      // Should be change to uint64_cu* 
    uint64_cu *d_PL, *d_startInputlist, *d_elementsPerILSplit, *d_PL_len;

    // Calculate memory sizes required:
    uint64_cu size_PL = (pheader.length) * sizeof(uint64_cu);
    uint64_cu elementsPerILSplit = IL_len / total_gpus;                               // WARNING: 'total_gpus' should be a power of 2 (code added for this check)

    // Calculate number of blocks (of 'int' type) required to store IL for a specific GPU (i.e. after splitting original IL):
    uint64_cu blocksFor_splitIL = (elementsPerILSplit / (sizeof(int) * 8));                 // Change the sizeof(param) to int / uint64_cu as per decision
    blocksFor_splitIL = (elementsPerILSplit % (sizeof(uint64_cu) * 8)) ? blocksFor_splitIL + 1 : blocksFor_splitIL;     // Taking ceiling of decimal (which will mean that last few bits will be unused by us)

    // Space for device copies:
    gpuErrchk( hipMalloc((void **) &d_IL, blocksFor_splitIL*sizeof(int)));
    gpuErrchk( hipMalloc((void **) &d_PL, size_PL));
    gpuErrchk( hipMalloc((void **) &d_startInputlist, sizeof(uint64_cu)) );
    //gpuErrchk( hipMalloc((void **) &d_blocksFor_splitIL, sizeof(uint64_cu)) );
    gpuErrchk( hipMalloc((void **) &d_elementsPerILSplit, sizeof(uint64_cu)) );
    //gpuErrchk( hipMalloc((void **) &d_ILlenPerGPU, sizeof(uint64_cu)) );
    gpuErrchk( hipMalloc((void **) &d_PL_len, sizeof(uint64_cu)) );

    // Calculate the start value of I/P list for kernel of current GPU:
    uint64_cu startInputlist = (gpu_id * elementsPerILSplit) + gpu_data.IL_start;                               

    // Copy the data to the device (GPU):
    gpuErrchk( hipMemcpy(d_PL, pheader.primelist, size_PL, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_startInputlist, &startInputlist, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    //gpuErrchk( hipMemcpy(d_blocksFor_splitIL, &blocksFor_splitIL, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_elementsPerILSplit, &elementsPerILSplit, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    //gpuErrchk( hipMemcpy(d_ILlenPerGPU, &ILlenPerGPU, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_PL_len, &PL_len, sizeof(uint64_cu), hipMemcpyHostToDevice) );

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Print and Cross-check the data received by this API:
    printf("Running on GPU with ID = %d\n", gpu_id);
    printf("size_PL = %llu bytes\n", size_PL);
    printf("IL_len = %llu\n", IL_len);
    printf("IL_start = %llu\n", gpu_data.IL_start);
    printf("IL_end = %llu\n\n", gpu_data.IL_end);
    printf("blocksFor_splitIL = %llu blocks of 'int' type\n", blocksFor_splitIL);

    printf("startInputlist = %llu\n\n", startInputlist);
    printf("elementsPerILSplit = %llu\n", elementsPerILSplit);
    printf("PL_len (= pheader.length) = %llu\n", PL_len);

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////


    // Launch the GPU kernel:
    // Should pass the 'blocksFor_splitIL' too?
    //prime_generator<<<(PL_len/THREADS_PER_BLOCK) + 1 , THREADS_PER_BLOCK>>>(d_IL, d_PL, d_startInputlist, d_blocksFor_splitIL, d_elementsPerILSplit);
    //
    cout << "Block Size: " <<  PL_len/THREADS_PER_BLOCK + 1 << endl;
    //cout << "Threads Per block: " << THREADS_PER_BLOCK << endl;                                                                                                      10^6-10^3/2         #168
    prime_generator<<<dim3((PL_len/THREADS_PER_BLOCK) + 1,1,1 ), dim3(THREADS_PER_BLOCK,1,1)>>>(d_IL, d_PL, d_startInputlist, d_elementsPerILSplit, d_PL_len);


    // Allocate space on host to copy back the splitIL from device:
    int *result; // = (int*) malloc(blocksFor_splitIL*sizeof(int));
    gpuErrchk(hipHostMalloc((void**)&result,blocksFor_splitIL*sizeof(int)));

    cout << "KTB: " << blocksFor_splitIL*sizeof(int) << " Result: " << result<<endl;
    // Copy the result back to the host:
    //    yellow_start();
    cout << "*********** Copying back IL: " << gpu_id << endl;
    //color_reset();
    //sleep(4);
    gpuErrchk( hipMemcpy(result, d_IL, blocksFor_splitIL*sizeof(int), hipMemcpyDeviceToHost) );
    cout <<endl<< "#################### END of gpu_id "<< gpu_id << " ####################"<<endl;

    uint64_cu foundPrimes = 0 ;
    for(uint64_cu index=0;index<elementsPerILSplit; index++){
        uint64_cu bucket = index / (WORD);
        uint64_cu setbit = index % (WORD);
       // uint64_cu actualNumber = startInputlist + index;
        if( !(result[bucket] & (1U << (setbit)))){
            //cout << actualNumber << " is prime?? "<< endl;
            foundPrimes ++;
        }
    }

    //red_start();
    cout << "*********** I am GPU: " << gpu_id << ", foundPrimes "<< foundPrimes << endl;

    // TODO : make it malloc
    uint64_cu* newPrimeList = new uint64_cu[foundPrimes];
    uint64_cu count = 0;
    for(uint64_cu index=0;index<elementsPerILSplit; index++){
        uint64_cu bucket = index / (WORD);
        uint64_cu setbit = index % (WORD);
        uint64_cu actualNumber = startInputlist + index;
        if( !(result[bucket] & (1U << (setbit)))){
            newPrimeList[count++] = actualNumber;
        }
    }

    //printList(newPrimeList,foundPrimes);
    //ThreadRetValue* tretvalue = (ThreadRetValue* ) malloc(sizeof(ThreadRetValue)); TODO: non issue of new against malloc
    ThreadRetValue* tretvalue = new ThreadRetValue();
    tretvalue->primelist = newPrimeList;
    tretvalue->length = foundPrimes;

    return tretvalue;

    //color_reset();
    // SOUMYADEEP :: Needs to make sure additional unused bits in IL (after ceiling) are converted to values other than 0, 
    // else they might be interpreted wrongly as primes:

    /*
    // Free GPU memory:
    hipFree(d_IL);
    hipFree(d_PL);
    hipFree(d_startInputlist);
    //hipFree(d_blocksFor_splitIL);
    hipFree(d_elementsPerILSplit);
    hipFree(d_PL_len);

     */
}



/* NOTES:
   1) Finalize the function parameters. They vary across APIs. (kernel launcher)
 */



/* NOTES:
   1) Finalize the function parameters. They vary across APIs. (kernel launcher)
 */






PrimeHeader calculate_primes_on_cpu(PrimeHeader pheader, uint64_cu pl_end_number ) {

    // Time Variables
    hipEvent_t start, stop;
    float time;
    gpuErrchk( hipEventCreate (&start));
    gpuErrchk( hipEventCreate (&stop));

    // Create Small 
    if (DEBUG >=2) {
        cout << "Allocating SMALL_SIEVE" << endl;
    }

    bool *small_sieve = new bool [pl_end_number];

    // Initialize Small Sieve
    for (uint64_cu i = 0; i < pl_end_number; i++) {
        small_sieve[i] = true;
    }

    // Compute Small Sieve on CPU
    hipEventRecord(start,0);

    for (uint64_cu i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (uint64_cu j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    gpuErrchk( hipEventRecord(stop,0));
    gpuErrchk( hipEventSynchronize(stop));
    gpuErrchk( hipEventElapsedTime(&time, start, stop));
    printf("CPU Time: %.2f ms till end prime number: %llu\n", time, pl_end_number);

    // Count Total Primes
    uint64_cu small_sieve_counter = 0;
    for (uint64_cu i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            //cout << i << " "; // To display prime numbers
            small_sieve_counter++;
        }
    }
    //cout << endl;

    if (DEBUG >= 1) {
        cout << "Total Primes Calculated on CPU: " << small_sieve_counter << endl;
    }

    //uint64_cu *prime_list = new uint64_cu [small_sieve_counter];
    uint64_cu *prime_list = (uint64_cu*) malloc(small_sieve_counter * sizeof(uint64_cu));

    // Storing numbers from the sieve to an array.
    uint64_cu inner_counter = 0;
    for (uint64_cu i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }
    pheader.primelist=prime_list;
    pheader.length=small_sieve_counter;
    pheader.lastMaxNo=pl_end_number; 
    writePrimes(pheader.primelist,pheader.length,pheader.lastMaxNo);

    return (pheader);
}

void writePrimes(uint64_cu primes[], uint64_cu length, uint64_cu lastNo){
    FILE * fout1 = fopen(PRIME_FILENAME,"ab+");
    if(!fout1){
        fprintf(stderr,"Error opening %s file for writing primes, error-> %s",PRIME_FILENAME,strerror(errno));
        exit(1);
    }

    PrimeHeader hdr;
    hdr.primelist = NULL;
    hdr.lastMaxNo = lastNo;
    hdr.length = length;

    size_t num = fwrite(&hdr, sizeof(PrimeHeader), 1, fout1);
    if(num!=1){
        fprintf(stdout,"Error writing prime header needed 1 , written only %ld",num);
        exit(1);
    }
    num = fwrite(primes, INTSIZE, length, fout1);
    if(num!=length){
        fprintf(stderr,"Error writing prime header needed %llu , written only %ld",length,num);
        exit(1);
    }
    num = fclose(fout1);
    if(num != 0){
        fprintf(stderr,"Error clossing %s file, error-> %s",PRIME_FILENAME,strerror(errno));
        exit(1);
    }
}

PrimeHeader readPrimes(){
    PrimeHeader ret;
    FILE* fin = fopen(PRIME_FILENAME,"rb");
    if(!fin){
        ret.lastMaxNo = 0 ;
        ret.length = 0;
        ret.primelist = NULL;
        printf("fin null pointer");
        return ret;
    }
    uint64_cu aggregatePrimes = 0 ;
    PrimeHeader hdr;
    uint64_cu offset = 0;
    printf("\nFIRST PASS: to find number of total primes\n");
    while(!feof(fin)){
        uint64_cu nread = fread(&hdr, sizeof(PrimeHeader), 1, fin);
        if(nread == 0)break;
        aggregatePrimes += hdr.length;
        printf("\nnread %llu ",nread);
        printf("\tlastMaxNo-> %llu ",hdr.lastMaxNo);
        printf("\tlength -> %llu ",hdr.length);
        // skip past primes in current line
        int ret = fseek(fin, (hdr.length *INTSIZE) , SEEK_CUR);
        if(ret==-1){
            fprintf(stderr,"Error in fseek %s file, error-> %s",PRIME_FILENAME,strerror(errno));
            exit(1);
        }
    }

    printf("\nAggregatePrimes %llu",aggregatePrimes);

    // now read all primes
    printf("\nSECOND PASS: to read all primes\n");
    fseek(fin,0,SEEK_SET);
    uint64_cu* retPtr = (uint64_cu*) malloc(aggregatePrimes * INTSIZE);
    if(!retPtr){
        fprintf(stderr,"Error in malloc of %llu primes, error-> %s",aggregatePrimes,strerror(errno));
        exit(1);
    }
    offset = 0;

    while(!feof(fin)){
        uint64_cu nread = fread(&hdr, sizeof(PrimeHeader), 1, fin);
        if(nread == 0)break;
        printf("\nnread %llu ",nread);
        ret.lastMaxNo = hdr.lastMaxNo;
        printf("\tlastMaxNo-> %llu ",hdr.lastMaxNo);
        printf("\tlength -> %llu ",hdr.length);
        uint64_cu nreadArr = fread(retPtr + offset ,INTSIZE,hdr.length,fin);
        if(nreadArr == 0){
            fprintf(stderr,"Error in reading of %llu primes, 0 were read",hdr.length);
            exit(1);
        }
        printf("\t %llu",nreadArr);
        offset += hdr.length;
    }
    printf("\n*************  PRINT AGGREGATE PRIMES ****************\n");
    //printList(retPtr,aggregatePrimes);
    ret.length = aggregatePrimes;
    ret.primelist = retPtr;
    size_t num = fclose(fin);
    if(num != 0){
        fprintf(stderr,"Error clossing %s file, error-> %s",PRIME_FILENAME,strerror(errno));
    }
    return ret;
}

