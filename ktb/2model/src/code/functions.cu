#include "hip/hip_runtime.h"
#include "functions.h"
#include "debugger.h"

extern PrimeHeader pheader;
extern GpuHandler gpu_data;


using namespace std;

#define WARNINGS 0

long find_number_of_gpus() {
    // System command to find number of GPUs attached 
    // find /proc/driver/nvidia/gpus -type d | wc -l

    char cmd[100] = "find /proc/driver/nvidia/gpus -type d | wc -l\0";
    array<char, 128> buffer;
    string result;
    shared_ptr<FILE> pipe(popen(cmd, "r"), pclose);
    if (!pipe) throw runtime_error("popen() failed!");
    while (!feof(pipe.get())) {
        if (fgets(buffer.data(), 128, pipe.get()) != nullptr)
            result += buffer.data();
    }
    long number_of_gpus = (long)stoi(result);
    number_of_gpus--; // The systems command returns a value which is
    // one more than the actual number of GPUs.
    return (number_of_gpus);

    // Function Complete --KTB
}




void start_info() {
    // Will print all the stats about the program like
    // Number of GPU being used.
    // End Number being Calculated, and so on.

    green_start();
    cout << "\n\n\n\n\n\n\n\n\n\nProgram Start\n";
    color_reset();

}



void end_info() {
    // Statistics about the program goes here


    cout << endl<< endl<< endl;
}



void iteration_info() {
    // Statistics about every iteration of the program goes here


}

void kernelLauncher(int gpu_id) {
    /*
    Convention for naming variables:
    len: relates to number of elements
    size: relates to size of memory
    */

    uint64_cu IL_len =  gpu_data.IL_end - gpu_data.IL_start + 1;
    int total_gpus = gpu_data.gpus;
    uint64_cu PL_len = pheader.length;

    // Declare all pointers to GPU:
    int *d_IL;                                                                      // Should be change to uint64_cu* 
    uint64_cu *d_PL, *d_startInputlist, *d_elementsPerILSplit, *d_PL_len;

    // Calculate memory sizes required:
    uint64_cu size_PL = (pheader.length) * sizeof(uint64_cu);
    uint64_cu elementsPerILSplit = IL_len / total_gpus;                               // WARNING: 'total_gpus' should be a power of 2 (code added for this check)
    
    // Calculate number of blocks (of 'int' type) required to store IL for a specific GPU (i.e. after splitting original IL):
    uint64_cu blocksFor_splitIL = (elementsPerILSplit / (sizeof(int) * 8));                 // Change the sizeof(param) to int / uint64_cu as per decision
    blocksFor_splitIL = (elementsPerILSplit % (sizeof(uint64_cu) * 8)) ? blocksFor_splitIL + 1 : blocksFor_splitIL;     // Taking ceiling of decimal (which will mean that last few bits will be unused by us)

    // Space for device copies:
    gpuErrchk( hipMalloc((void **) &d_IL, blocksFor_splitIL*sizeof(int)));
    gpuErrchk( hipMalloc((void **) &d_PL, size_PL));
    gpuErrchk( hipMalloc((void **) &d_startInputlist, sizeof(uint64_cu)) );
    //gpuErrchk( hipMalloc((void **) &d_blocksFor_splitIL, sizeof(uint64_cu)) );
    gpuErrchk( hipMalloc((void **) &d_elementsPerILSplit, sizeof(uint64_cu)) );
    //gpuErrchk( hipMalloc((void **) &d_ILlenPerGPU, sizeof(uint64_cu)) );
    gpuErrchk( hipMalloc((void **) &d_PL_len, sizeof(uint64_cu)) );

    // Calculate the start value of I/P list for kernel of current GPU:
    uint64_cu startInputlist = (gpu_id * elementsPerILSplit) + gpu_data.IL_start;                               

    // Copy the data to the device (GPU):
    gpuErrchk( hipMemcpy(d_PL, pheader.primelist, size_PL, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_startInputlist, &startInputlist, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    //gpuErrchk( hipMemcpy(d_blocksFor_splitIL, &blocksFor_splitIL, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_elementsPerILSplit, &elementsPerILSplit, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    //gpuErrchk( hipMemcpy(d_ILlenPerGPU, &ILlenPerGPU, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_PL_len, &PL_len, sizeof(uint64_cu), hipMemcpyHostToDevice) );

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Print and Cross-check the data received by this API:
    printf("Running on GPU with ID = %d\n", gpu_id);
    printf("gpus = %llu\n", gpu_data.gpus);
    printf("size_PL = %llu bytes\n", size_PL);
    printf("IL_len = %llu\n", IL_len);
    printf("IL_start = %llu\n", gpu_data.IL_start);
    printf("IL_end = %llu\n\n", gpu_data.IL_end);
    printf("blocksFor_splitIL = %llu blocks of 'int' type\n", blocksFor_splitIL);

    printf("startInputlist = %llu\n\n", startInputlist);
    printf("elementsPerILSplit = %llu\n", elementsPerILSplit);
    printf("PL_len (= pheader.length) = %llu\n", PL_len);
   
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////

    // Select the device:
    gpuErrchk( hipSetDevice(gpu_id) );

    // Launch the GPU kernel:
    // Should pass the 'blocksFor_splitIL' too?
    //prime_generator<<<(PL_len/THREADS_PER_BLOCK) + 1 , THREADS_PER_BLOCK>>>(d_IL, d_PL, d_startInputlist, d_blocksFor_splitIL, d_elementsPerILSplit);
    //                                                                                                        10^6-10^3/2         #168
    prime_generator<<<(PL_len/THREADS_PER_BLOCK) + 1 , THREADS_PER_BLOCK>>>(d_IL, d_PL, d_startInputlist, d_elementsPerILSplit, d_PL_len);
    

    // Allocate space on host to copy back the splitIL from device:
    int *result = (int*) malloc(blocksFor_splitIL*sizeof(int));
cout << "KTB: " << blocksFor_splitIL*sizeof(int) << "Result: " << result<<endl;
    // Copy the result back to the host:
    gpuErrchk( hipMemcpy(result, d_IL, blocksFor_splitIL*sizeof(int), hipMemcpyDeviceToHost) );

/*  ********** DECODING: NOT WORKING FOR NOW **************
    for(uint64_cu i=0; i<blocksFor_splitIL; i++) {
        int bitvec = result[i];
        int num = 1;
        for(int j=sizeof(int)*8; j>0; j--) {
            int value = bitvec & num ;
            if(value == num) {
                printf("%llu  ", (uint64_cu) (((uint64_cu)sizeof(int)*(uint64_cu)i*8) + (uint64_cu) j) );
                printf("Hi  ");
            }
            num = num << 1;
        }
        printf("\n");
    }
*/

    // Printing the elements of IL after GPU computation as it is (not decoding to get the prime):
//      if (gpu_id==1) {
//      for(int i=0; i<blocksFor_splitIL; i++) {
//          if (result[i] != 0)
//          printf("%d  ", result[i]);
//      }
//  }
red_start();
 cout << "*********** I am GPU: " << gpu_id << endl;
color_reset();
// SOUMYADEEP :: Needs to make sure additional unused bits in IL (after ceiling) are converted to values other than 0, 
// else they might be interpreted wrongly as primes:


    // Free GPU memory:
    hipFree(d_IL);
    hipFree(d_PL);
    hipFree(d_startInputlist);
    //hipFree(d_blocksFor_splitIL);
    hipFree(d_elementsPerILSplit);
    hipFree(d_PL_len);
}



/* NOTES:
1) Finalize the function parameters. They vary across APIs. (kernel launcher)
*/



/* NOTES:
1) Finalize the function parameters. They vary across APIs. (kernel launcher)
*/






PrimeHeader calculate_primes_on_cpu(PrimeHeader pheader, uint64_cu pl_end_number ) {

// Time Variables
hipEvent_t start, stop;
float time;
gpuErrchk( hipEventCreate (&start));
gpuErrchk( hipEventCreate (&stop));



// Create Small 
if (DEBUG >=2) {
    cout << "Allocating SMALL_SIEVE" << endl;
}



bool *small_sieve = new bool [pl_end_number];


    
// Initialize Small Sieve
for (uint64_cu i = 0; i < pl_end_number; i++) {
    small_sieve[i] = true;
}

    // Compute Small Sieve on CPU
    hipEventRecord(start,0);
    
    for (uint64_cu i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (uint64_cu j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    gpuErrchk( hipEventRecord(stop,0));
    gpuErrchk( hipEventSynchronize(stop));
    gpuErrchk( hipEventElapsedTime(&time, start, stop));
    printf("CPU Time: %.2f ms till end prime number: %llu\n", time, pl_end_number);


    // Count Total Primes
    uint64_cu small_sieve_counter = 0;
    for (uint64_cu i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            //cout << i << " "; // To display prime numbers
            small_sieve_counter++;
        }
    }
    //cout << endl;

    if (DEBUG >= 1) {
        cout << "Total Primes Calculated on CPU: " << small_sieve_counter << endl;
    }

    uint64_cu *prime_list = new uint64_cu [small_sieve_counter];

    // Storing numbers from the sieve to an array.
    uint64_cu inner_counter = 0;
    for (uint64_cu i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }
    pheader.primelist=prime_list;
    pheader.length=small_sieve_counter;
    pheader.lastMaxNo=pl_end_number; 

    return (pheader);
}
