#include "hip/hip_runtime.h"
#include "functions.h"
#include "debugger.h"

extern PrimeHeader pheader;
extern GpuHandler gpu_data;


using namespace std;

#define WARNINGS 0

long find_number_of_gpus() {
    // System command to find number of GPUs attached 
    // find /proc/driver/nvidia/gpus -type d | wc -l

    char cmd[100] = "find /proc/driver/nvidia/gpus -type d | wc -l\0";
    array<char, 128> buffer;
    string result;
    shared_ptr<FILE> pipe(popen(cmd, "r"), pclose);
    if (!pipe) throw runtime_error("popen() failed!");
    while (!feof(pipe.get())) {
        if (fgets(buffer.data(), 128, pipe.get()) != nullptr)
            result += buffer.data();
    }
    long number_of_gpus = (long)stoi(result);
    number_of_gpus--; // The systems command returns a value which is
    // one more than the actual number of GPUs.
    return (number_of_gpus);

    // Function Complete --KTB
}



void start_info() {
    // Will print all the stats about the program like
    // Number of GPU being used.
    // End Number being Calculated, and so on.

    green_start();
    cout << "\n\n\n\n\n\n\n\n\n\nProgram Start\n";
    color_reset();

}



void end_info() {
    // Statistics about the program goes here


    cout << endl<< endl<< endl;
}



void iteration_info() {
    // Statistics about every iteration of the program goes here


}

// Launch the kernel:

void kernelLauncher(int gpu_id) {

    
    
    uint64_cu Il_len =  gpu_data->IL_end - gpu_data->IL_start;
    
    int total_gpus=gpu_data->gpus;
	// Round off the number of GPUs to the next lower power of 2:
	// for(int i=0; i<10; i++) {
	// 	if(pow(2,i) > gpu_data->gpus)
    //         total_gpus = pow(2, i-1);
	// 		break;
	// }

    int *d_IL = NULL;
    uint64_cu *d_PL = NULL;
    uint64_cu *d_startPrimelist;
    uint64_cu *d_splitILsize;
    uint64_cu *d_elementsPerSplit;

	// Select the device:
	gpuErrchk( hipSetDevice(gpu_id) );

	// Calculate memory sizes required:
	uint64_cu elementsPerSplit = IL_len / total_gpus;			// WARNING: 'total_gpus' should be a power of 2 (code added for this check)
	uint64_cu splitILsize = (elementsPerSplit / (sizeof(uint64_cu) * 8)); 				// Confirm during code integration whether a '+1' is needed in the end.
	uint64_cu size_PL = (pheader->length) * sizeof(uint64_cu);

	// Space for device copies:
	gpuErrchk( hipMalloc((void **) &d_IL, splitILsize));
	gpuErrchk( hipMalloc((void **) &d_PL, size_PL));
    gpuErrchk( hipMalloc((void **) &d_startPrimelist, sizeof(uint64_cu)) );
    gpuErrchk( hipMalloc((void **) &d_splitILsize, sizeof(uint64_cu)) );
    gpuErrchk( hipMalloc((void **) &d_elementsPerSplit, sizeof(uint64_cu)) );

    // Calculate the start value of I/P list for kernel of current GPU:
    uint64_cu c_startPrimelist = gpu_id * elementsPerSplit;                                // uint64_cu conflict

    
	// Copy the data to the device (GPU):
	gpuErrchk( hipMemcpy(d_PL, pheader->primelist, size_PL, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_startPrimelist, &c_startPrimelist, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_splitILsize, &splitILsize, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_elementsPerSplit, &elementsPerSplit, sizeof(uint64_cu), hipMemcpyHostToDevice) );

    

	// Launch the GPU kernel:
	prime_generator<<<(PL_len/THREADS_PER_BLOCK) + 1 , THREADS_PER_BLOCK>>>(d_IL, d_PL, d_startPrimelist, d_splitILsize, d_elementsPerSplit);

}



/* NOTES:
1) Finalize the function parameters. They vary across APIs. (kernel launcher)
*/
