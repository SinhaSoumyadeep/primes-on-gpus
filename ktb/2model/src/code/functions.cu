#include "hip/hip_runtime.h"
#include "functions.h"
#include "debugger.h"

extern PrimeHeader pheader;
extern GpuHandler gpu_data;


using namespace std;

#define WARNINGS 0

long find_number_of_gpus() {
    // System command to find number of GPUs attached 
    // find /proc/driver/nvidia/gpus -type d | wc -l

    char cmd[100] = "find /proc/driver/nvidia/gpus -type d | wc -l\0";
    array<char, 128> buffer;
    string result;
    shared_ptr<FILE> pipe(popen(cmd, "r"), pclose);
    if (!pipe) throw runtime_error("popen() failed!");
    while (!feof(pipe.get())) {
        if (fgets(buffer.data(), 128, pipe.get()) != nullptr)
            result += buffer.data();
    }
    long number_of_gpus = (long)stoi(result);
    number_of_gpus--; // The systems command returns a value which is
    // one more than the actual number of GPUs.
    return (number_of_gpus);

    // Function Complete --KTB
}



void start_info() {
    // Will print all the stats about the program like
    // Number of GPU being used.
    // End Number being Calculated, and so on.

    green_start();
    cout << "\n\n\n\n\n\n\n\n\n\nProgram Start\n";
    color_reset();

}



void end_info() {
    // Statistics about the program goes here


    cout << endl<< endl<< endl;
}



void iteration_info() {
    // Statistics about every iteration of the program goes here


}

// Launch the kernel:

void kernelLauncher(int gpu_id) {

    
    
    uint64_cu IL_len =  gpu_data.IL_end - gpu_data.IL_start;
    
    int total_gpus=gpu_data.gpus;
	// Round off the number of GPUs to the next lower power of 2:
	// for(int i=0; i<10; i++) {
	// 	if(pow(2,i) > gpu_data->gpus)
    //         total_gpus = pow(2, i-1);
	// 		break;
	// }

    int *d_IL = NULL;
    uint64_cu *d_PL = NULL;
    uint64_cu *d_startPrimelist;
    uint64_cu *d_splitILsize;
    uint64_cu *d_elementsPerSplit;

	// Select the device:
	gpuErrchk( hipSetDevice(gpu_id) );

	// Calculate memory sizes required:
	uint64_cu elementsPerSplit = IL_len / total_gpus;			// WARNING: 'total_gpus' should be a power of 2 (code added for this check)
	uint64_cu splitILsize = (elementsPerSplit / (sizeof(uint64_cu) * 8)); 				// Confirm during code integration whether a '+1' is needed in the end.
	uint64_cu size_PL = (pheader.length) * sizeof(uint64_cu);

	// Space for device copies:
	gpuErrchk( hipMalloc((void **) &d_IL, splitILsize));
	gpuErrchk( hipMalloc((void **) &d_PL, size_PL));
    gpuErrchk( hipMalloc((void **) &d_startPrimelist, sizeof(uint64_cu)) );
    gpuErrchk( hipMalloc((void **) &d_splitILsize, sizeof(uint64_cu)) );
    gpuErrchk( hipMalloc((void **) &d_elementsPerSplit, sizeof(uint64_cu)) );

    // Calculate the start value of I/P list for kernel of current GPU:
    uint64_cu c_startPrimelist = gpu_id * elementsPerSplit;                                // uint64_cu conflict

    
	// Copy the data to the device (GPU):
	gpuErrchk( hipMemcpy(d_PL, pheader.primelist, size_PL, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_startPrimelist, &c_startPrimelist, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_splitILsize, &splitILsize, sizeof(uint64_cu), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_elementsPerSplit, &elementsPerSplit, sizeof(uint64_cu), hipMemcpyHostToDevice) );

    uint64_cu PL_len = pheader.length;

    // Launch the GPU kernel:
    cout << "splitILsize: "<< splitILsize << endl;
    cout << "elementsPerSplit: "<< elementsPerSplit << endl;
    cout << "c_startPrimelist: "<< c_startPrimelist << endl;
 
    
    
    cout << "d_IL: "<< d_IL << endl;
    cout << "d_PL: "<< d_PL << endl;
    cout << "d_startPrimelist: "<< d_startPrimelist << endl;
    cout << "d_splitILsize: "<< d_splitILsize << endl;
    cout << "d_elementsPerSplit: "<< d_elementsPerSplit << endl;
    
    

	prime_generator<<<(PL_len/THREADS_PER_BLOCK) + 1 , THREADS_PER_BLOCK>>>(d_IL, d_PL, d_startPrimelist, d_splitILsize, d_elementsPerSplit);

}



/* NOTES:
1) Finalize the function parameters. They vary across APIs. (kernel launcher)
*/



PrimeHeader calculate_primes_on_cpu(PrimeHeader pheader, uint64_cu pl_end_number, ) {
// Time Variables
hipEvent_t start, stop;
float time;
gpuErrchk(hipEventCreate (&start));
gpuErrchk(hipEventCreate (&stop));



// Create Small 
if (DEBUG >=2) {
    cout << "Allocating SMALL_SIEVE" << endl;
}



bool *small_sieve = new bool [pl_end_number];


    
// Initialize Small Sieve
for (uint64_cu i = 0; i < pl_end_number; i++) {
    small_sieve[i] = true;
}

    // Compute Small Sieve on CPU
    hipEventRecord(start,0);
    
    for (uint64_cu i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (uint64_cu j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    gpuErrchk( hipEventRecord(stop,0));
    gpuErrchk( hipEventSynchronize(stop));
    gpuErrchk( hipEventElapsedTime(&time, start, stop));
    printf("CPU Time: %.2f ms till end prime number: %llu\n", time, pl_end_number);


    // Count Total Primes
    uint64_cu small_sieve_counter = 0;
    for (uint64_cu i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            //cout << i << " "; // To display prime numbers
            small_sieve_counter++;
        }
    }
    //cout << endl;

    total_primes += small_sieve_counter;
    if (DEBUG >= 1) {
        cout << "Total Primes Calculated on CPU: " << small_sieve_counter << endl;
    }

    uint64_cu *prime_list = new uint64_cu [small_sieve_counter];

    // Storing numbers from the sieve to an array.
    uint64_cu inner_counter = 0;
    for (uint64_cu i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }
    pheader.primelist=prime_list;
    pheader.length=small_sieve_counter;
    pheader.lastMaxNo=pl_end_number;

}