#include "functions.h"
#include "debugger.h"

using namespace std;

#define WARNINGS 0



 inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
 {
    if (code != hipSuccess) 
    {
 
     fprintf(stderr,"\e[1;31mGPUassert: %s %s %d \e[0m\n", hipGetErrorString(code), file, line);
 
       if (abort) exit(code);
    }
 }