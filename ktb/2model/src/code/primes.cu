#include "hip/hip_runtime.h"
#include <functions.h>
#include <debugger.h>

using namespace std;


// Global Variables.
uint64_cu pl_end_number = 10000;
int number_of_gpus = 1;

PrimeHeader pheader;
GpuHandler gpu_data;
//long long int end_val = 1000000;




// ********************** KERNEL DEFINITION **********************

__global__ void prime_generator(int* d_input_list, uint64_cu* d_prime_list, uint64_cu* d_startPrimelist,uint64_cu* d_total_inputsize,uint64_cu* d_number_of_primes)
{
 
uint64_cu tid = (blockIdx.x*blockDim.x) + threadIdx.x;

     
if (tid < *d_number_of_primes) {
    printf("Kaustubh\n");
    uint64_cu primes=d_prime_list[tid];
    for(uint64_cu i=0;i<=d_total_inputsize[0];i++) { // Added less than eual to here.
        uint64_cu bucket= i/(WORD);
        uint64_cu setbit= i%(WORD);
        uint64_cu number=d_startPrimelist[0]+i;
        if(number%primes==0) {
            //printf("%llu is divisible by %llu \n", number,primes);
            d_input_list[bucket]=d_input_list[bucket]| 1U<<setbit;
            }
        }
    }
}



// ********************** PTHREAD ITERATION **********************

void *one_iteration(void *tid) {
    long gpu_id = (long) tid; // Dont use tid, Use gpu_id instead
    if (DEBUG >= 1) {
        cout << "Launched GPU Handler: " << gpu_id << endl;
    }

    hipEvent_t start, stop;
    

    

// Saurin's Code
gpu_data.IL_start = pl_end_number+1;
gpu_data.IL_end = pl_end_number*pl_end_number;
kernelLauncher(gpu_id);

}




// ********************** MAIN FUNCTION **********************

int main(int argc, char *argv[]) { 

    start_info(); // Complete

    number_of_gpus = find_number_of_gpus(); // Complete
    number_of_gpus = pow(2,int(log(number_of_gpus)/log(2)));
    gpu_data.gpus = number_of_gpus;

    // Accepting input from Console
    switch (argc) { // For getting input from console
        case 6:
            //long input_5;
            //input_5 = atol(argv[5]); //Fifth Input
            
        case 5:
            //long input_4;
            //input_4 = atol(argv[4]); //Fourth Input
            
        case 4:
            //long input_3;
            //input_3 = atol(argv[3]); // Third Input
            
        case 3:
            long input_2;
            input_2 = atol(argv[2]); // Second Input
            number_of_gpus = (int)input_2; // Number of GPUs on the NODE.
            // Over-ride with input value.
        case 2:
            long input_1;
            input_1 = atol(argv[1]); // First input
            pl_end_number = (uint64_cu)input_1;

            break;
        case 1:
            // Keep this empty
            break;
        default:
            red_start();
            cout << "FATAL ERROR: Wrong Number of Inputs" << endl; // If incorrect number of inputs are used.
            color_reset();
            return 1;
    }

    if (number_of_gpus != find_number_of_gpus()) {
        cyan_start();
        cout << "INFO: Running on " << number_of_gpus << " GPUs out of " << find_number_of_gpus() << " GPUs." << endl;
        color_reset();
    }

    pheader = calculate_primes_on_cpu(pheader,pl_end_number); 

//    while(end_reached) {

    //  *************** PTHREADS LAUNCH *******************


    pthread_t *thread = new pthread_t [number_of_gpus];
    int *thread_error = new int [number_of_gpus];

    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_create(&thread[i], NULL, one_iteration, (void *) i);
        if (thread_error[i] && WARNINGS) {
            yellow_start();
            cout << "Warning: Thread " << i << " failed to launch" << endl;
            cout << "GPU: " << i << " is being mishandled." << endl;
            color_reset();
        }
    }
    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_join(thread[i], NULL);
    }

// output_combine();

    // INLINE
    //iteration_info();

//}


// CODE

    // INLINE
    //end_info();

    return 0;
}

