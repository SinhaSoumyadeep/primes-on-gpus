#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <functions.h>
#include <debugger.h>

using namespace std;

#define block_size   32
#define vector_size 1000
#define DEBUG 1

__global__ void prime( int *a, int *b, int *c ) {
    int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

    if (tid < vector_size){
        c[tid] = a[tid] + b[tid];                   // add vectors together                
    }
}



// ********************** MAIN FUNCTION **********************

unsigned long long int pl_end_number = 1000;
unsigned long long int end_val = 1000;


int main(int argc, char *argv[]) { 

    

    green_start();
    cout << "\n\n\n\n\n\n\n\n\n\nProgram Start\n";
    color_reset();
    
    // Accepting input from Console
    switch (argc) { // For getting input from console
        case 6:
            long input_5;
            input_5 = atol(argv[5]); //Fifth Input
            //num_threads = input_5;
        case 5:
            long input_4;
            input_4 = atol(argv[4]); //Fourth Input
            //in_parallel = input_4;
        case 4:
            long input_3;
            input_3 = atol(argv[3]); // Third Input
            //display_bit = input_3;
        case 3:
            long input_2;
            input_2 = atol(argv[2]); // Second Input
        case 2:
            long input_1;
            input_1 = atol(argv[1]); // First input
            pl_end_number = (unsigned long long int)input_1;

            break;
        case 1:
            // Keep this empty
            break;
        default:
            red_start();
            cout << "FATAL ERROR: Wrong Number of Inputs" << endl; // If incorrect number of inputs are used.
            color_reset();
            return 1;
    }








    // Select GPU
    hipSetDevice(1);

    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);


    // Create Small Sieve
    bool *small_sieve = new bool [pl_end_number];
    int bool_size = sizeof(bool);


    

    for (unsigned long long int i = 0; i < pl_end_number; i++) {
        small_sieve[i] = true;
    }

    hipEventRecord(start,0);
    
    for (unsigned long long int i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (unsigned long long int j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tPrime Numbers Computation Time on CPU: %.2f ms\n", time);

 //   cout << "Primes till 100\n";

    unsigned long long int small_sieve_counter = 0;
    for (unsigned long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            //cout << i << " ";
            small_sieve_counter++;
            //cout << small_sieve[i] << "    ";
        }
    }
    cout << endl;

    unsigned long long int *prime_list = new unsigned long long int [small_sieve_counter];

    unsigned long long int inner_counter = 0;
    for (unsigned long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }

    


    // Pointers in GPU memory
    int *dev_il;
    int *dev_pl;

    // Create Input list
    unsigned long long int start_number = pl_end_number+1;

    unsigned long long int il_size = pl_end_number*pl_end_number;

    bool *input_list = new bool [il_size];

    for (unsigned long long int i =0; i < il_size; i++) {
        input_list[i] = true;
    }

    
    // allocate the memory on the GPU
    //hipMalloc( (void**)&dev_il,  vector_size * bool_size );
    //hipMalloc( (void**)&dev_pl,  small_sieve_counter * bool_size );

    // copy the arrays 'a' and 'b' to the GPU
    // hipMemcpy( dev_a, a, vector_size * sizeof(int),
    //         hipMemcpyHostToDevice );
    // hipMemcpy( dev_b, b, vector_size * sizeof(int),
    //         hipMemcpyHostToDevice );


    //
    // GPU Calculation
    ////////////////////////

 //   printf("Running parallel job.\n");

    //int grid_size = (vector_size-1)/block_size;
    //grid_size++;

    //hipEventRecord(start,0);
    //prime<<<grid_size,block_size>>>( dev_a, dev_b, dev_c);

    //hipEventRecord(stop,0);
    //hipEventSynchronize(stop);

    //hipEventElapsedTime(&time, start, stop);
 //   printf("\tParallel Job Time: %.2f ms\n", time);

    // copy the array 'c' back from the GPU to the CPU
    // hipMemcpy( c_gpu, dev_c, vector_size * sizeof(int), 
    //         hipMemcpyDeviceToHost );

    // compare the results
    // int error = 0;
    // for (int i = 0; i < vector_size; i++) {
    //     if (c_cpu[i] != c_gpu[i]){
    //         error = 1;
    //         // printf( "Error starting element %d, %d != %d\n", i, c_gpu[i], c_cpu[i] );    
    //     }
    //     if (error) break; 
    // }

    // if (error == 0){
    //     printf ("Correct result. No errors were found.\n");
    // }

    // free the memory allocated on the GPU
    // hipFree( dev_a );
    // hipFree( dev_b );
    // hipFree( dev_c );

    // free(a);
    // free(b);
    // free(c_cpu);
    // free(c_gpu);

    return 0;
}

