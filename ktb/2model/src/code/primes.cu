#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <functions.h>
#include <debugger.h>

using namespace std;

#define block_size   32
#define DEBUG 1


// ********************** KERNEL DEFINITION **********************

__global__ void prime( bool *il, 
    long long int *pl, 
    long long int *dev_input_size_ptr, 
    long long int *dev_prime_size_ptr, 
    long long int *dev_pl_end_number_ptr ) {
    
        long long int dev_input_size = *dev_input_size_ptr;
        long long int dev_prime_size = *dev_prime_size_ptr; 
        long long int dev_pl_end_number = *dev_pl_end_number_ptr;



        long long int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id


    if (tid <= dev_prime_size) {
        long long int tpno = pl[tid];
        //printf("\tTID: %d", tid);
            for (long long int k=dev_pl_end_number; k<dev_input_size; k++) {
                if (k % tpno == 0) {
                    il[k] = false;                   // add vectors together                
            }
        }
    }
}




// Global Variables.
long long int pl_end_number = 1000;
long long int total_primes=0;
//long long int end_val = 1000000;


// ********************** MAIN FUNCTION **********************

int main(int argc, char *argv[]) { 


    hipProfilerStart();

    green_start();
    cout << "\n\n\n\n\n\n\n\n\n\nProgram Start\n";
    color_reset();
    
    // Accepting input from Console
    switch (argc) { // For getting input from console
        case 6:
            long input_5;
            input_5 = atol(argv[5]); //Fifth Input
            //num_threads = input_5;
        case 5:
            long input_4;
            input_4 = atol(argv[4]); //Fourth Input
            //in_parallel = input_4;
        case 4:
            long input_3;
            input_3 = atol(argv[3]); // Third Input
            //display_bit = input_3;
        case 3:
            long input_2;
            input_2 = atol(argv[2]); // Second Input
        case 2:
            long input_1;
            input_1 = atol(argv[1]); // First input
            pl_end_number = (long long int)input_1;

            break;
        case 1:
            // Keep this empty
            break;
        default:
            red_start();
            cout << "FATAL ERROR: Wrong Number of Inputs" << endl; // If incorrect number of inputs are used.
            color_reset();
            return 1;
    }








    // Select GPU
    hipSetDevice(1);

    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);


    // Create Small Sieve
    bool *small_sieve = new bool [pl_end_number];


    
    // Initialize Small Sieve
    for (long long int i = 0; i < pl_end_number; i++) {
        small_sieve[i] = true;
    }

    // Compute Small Sieve on CPU
    hipEventRecord(start,0);
    
    for (long long int i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (long long int j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("CPU Time: %.2f ms\n", time);


    // Count Total Primes
    long long int small_sieve_counter = 0;
    for (long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            // To display prime numbers
            //cout << i << " ";
            small_sieve_counter++;
        }
    }
    cout << endl;

    total_primes += small_sieve_counter;
    if (DEBUG >= 1) {
        cout << "Total Primes in Small Sieve: " << small_sieve_counter << endl;
    }

    long long int *prime_list = new long long int [small_sieve_counter];

    // Storing numbers from the sieve to an array.
    long long int inner_counter = 0;
    for (long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }

    
    // Create Input list on CPU
    long long int il_size = pl_end_number*pl_end_number;
    bool *input_list = new bool [il_size];
    for (long long int i =0; i < il_size; i++) {
        input_list[i] = true;
    }

    printf("Input List Size on CPU: %llu\n", il_size);



    // Pointers in GPU memory
    bool *dev_il;
    long long int *dev_pl;
    long long int *dev_input_size;
    long long int *dev_prime_size;
    long long int *dev_pl_end_number;   
    

    // Allocate the memory on the GPU
    hipMalloc( (void**)&dev_il,  il_size * sizeof(bool) );
    hipMalloc( (void**)&dev_pl,  small_sieve_counter * sizeof(long long int) );
    hipMalloc( (void**)&dev_input_size,  sizeof(long long int) );
    hipMalloc( (void**)&dev_prime_size,  sizeof(long long int) );
    hipMalloc( (void**)&dev_prime_size,  sizeof(long long int) );
    hipMalloc( (void**)&dev_pl_end_number,  sizeof(long long int) );


    // Copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_il, input_list, il_size * sizeof(bool),
             hipMemcpyHostToDevice );
    hipMemcpy( dev_pl, prime_list, small_sieve_counter * sizeof(long long int),
             hipMemcpyHostToDevice );
    hipMemcpy( dev_prime_size, &small_sieve_counter, sizeof(long long int),
             hipMemcpyHostToDevice );
    hipMemcpy( dev_input_size, &il_size, sizeof(long long int),
             hipMemcpyHostToDevice );
    hipMemcpy( dev_pl_end_number, &pl_end_number, sizeof(long long int),
             hipMemcpyHostToDevice );


    //
    // GPU Calculation
    ////////////////////////

 //   printf("Running parallel job.\n");

    int grid_size = (small_sieve_counter-1)/block_size;
    grid_size++;

    if (DEBUG >=1) {
        cout << "Grid Size: " << grid_size << endl;
        cout << "Block Size: " << block_size << endl;
        
    }


    // ********************** KERNEL LAUNCH **********************

    hipEventRecord(start,0);
    prime<<<grid_size,block_size>>>(dev_il, dev_pl, dev_input_size, dev_prime_size, dev_pl_end_number);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    printf("GPU Time: %.2f ms\n", time);

        // Create Output list on CPU
        bool *output_list = new bool [il_size];
        

    // copy the array Input List back from the GPU to the CPU
     hipMemcpy( output_list, dev_il, il_size * sizeof(bool), 
             hipMemcpyDeviceToHost );


    // Check Returned Primes
    long long int ret_primes=0;
    
    for (long long int i = pl_end_number; i < pl_end_number*pl_end_number; i++) {
        if (output_list[i] == true) {
            // To display prime numbers
            //cout << i << " ";
            ret_primes++;
            //small_sieve_counter++;
        }
    }

    total_primes += ret_primes; 
    cout << "Total Primes: "<< total_primes;
    cout << endl;
             
    

    // Free the memory allocated on the GPU
    hipFree( dev_il );
    hipFree( dev_pl );

    // free(a);
    // free(b);
    // free(c_cpu);
    // free(c_gpu);

    hipProfilerStop();

    return 0;
}

