#include "hip/hip_runtime.h"
#include <functions.h>
#include <debugger.h>

using namespace std;

#define block_size   32
#define DEBUG 1
#define GPU 0
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


// ********************** KERNEL DEFINITION **********************

__global__ void prime( bool *il, 
    long long int *pl, 
    long long int *dev_input_size_ptr, 
    long long int *dev_prime_size_ptr, 
    long long int *dev_pl_end_number_ptr ) {
    
        long long int dev_input_size = *dev_input_size_ptr;
        long long int dev_prime_size = *dev_prime_size_ptr; 
        long long int dev_pl_end_number = *dev_pl_end_number_ptr;



        long long int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id


    if (tid <= dev_prime_size) {
        long long int tpno = pl[tid];
        //printf("\tTID: %d", tid);
            for (long long int k=dev_pl_end_number; k<dev_input_size; k++) {
                if (k % tpno == 0) {
                    il[k] = false;                   // add vectors together                
            }
        }
    }
}


// ********************** PTHREAD LAUNCH **********************

void *one_iteration(void *tid) {
    long thread_id = (long) tid;

    cout << "I am thread " << thread_id << endl;

}




// Global Variables.
long long int pl_end_number = 1000;
long long int total_primes=0;
long number_of_gpus = 1;
//long long int end_val = 1000000;


// ********************** MAIN FUNCTION **********************

int main(int argc, char *argv[]) { 


 
    green_start();
    cout << "\n\n\n\n\n\n\n\n\n\nProgram Start\n";
    color_reset();
    
    // Accepting input from Console
    switch (argc) { // For getting input from console
        case 6:
            //long input_5;
            //input_5 = atol(argv[5]); //Fifth Input
            
        case 5:
            //long input_4;
            //input_4 = atol(argv[4]); //Fourth Input
            
        case 4:
            //long input_3;
            //input_3 = atol(argv[3]); // Third Input
            
        case 3:
            long input_2;
            input_2 = atol(argv[2]); // Second Input
            number_of_gpus = (int)input_2; // Number of GPUs on the NODE.
        case 2:
            long input_1;
            input_1 = atol(argv[1]); // First input
            pl_end_number = (long long int)input_1;

            break;
        case 1:
            // Keep this empty
            break;
        default:
            red_start();
            cout << "FATAL ERROR: Wrong Number of Inputs" << endl; // If incorrect number of inputs are used.
            color_reset();
            return 1;
    }




    // Pthreads Launch
    pthread_t *thread = new pthread_t [number_of_gpus];
    int *thread_error = new int [number_of_gpus];

    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_create(&thread[i], NULL, one_iteration, (void *) i);
        if (thread_error[i]) {
            yellow_start();
            cout << "Warning: Thread " << i << " failed to launch" << endl;
            cout << "GPU: " << i << " is being mishandled." << endl;
            color_reset();
        }
    }
    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_join(thread[i], NULL);
    }


    // Select GPU
    gpuErrchk(hipSetDevice(GPU));

    // Time Variables
    hipEvent_t start, stop;
    float time;
    gpuErrchk(hipEventCreate (&start));
    gpuErrchk(hipEventCreate (&stop));


    // Create Small 
    if (DEBUG >=1) {
        cout << "Allocating SMALL_SIEVE" << endl;
    }
    bool *small_sieve = new bool [pl_end_number];


    
    // Initialize Small Sieve
    for (long long int i = 0; i < pl_end_number; i++) {
        small_sieve[i] = true;
    }

    // Compute Small Sieve on CPU
    hipEventRecord(start,0);
    
    for (long long int i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (long long int j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    gpuErrchk( hipEventRecord(stop,0));
    gpuErrchk( hipEventSynchronize(stop));
    gpuErrchk( hipEventElapsedTime(&time, start, stop));
    printf("CPU Time: %.2f ms\n", time);


    // Count Total Primes
    long long int small_sieve_counter = 0;
    for (long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            // To display prime numbers
            //cout << i << " ";
            small_sieve_counter++;
        }
    }
    cout << endl;

    total_primes += small_sieve_counter;
    if (DEBUG >= 1) {
        cout << "Total Primes in Small Sieve: " << small_sieve_counter << endl;
    }

    if (DEBUG >=1) {
        cout << "Allocating PRIME_LIST" << endl;
    }
    long long int *prime_list = new long long int [small_sieve_counter];

    // Storing numbers from the sieve to an array.
    long long int inner_counter = 0;
    for (long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }

    
    // Create Input list on CPU
    long long int il_size = pl_end_number*pl_end_number;
    if (DEBUG >=1) {
        cout << "Allocating INPUT_LIST" << endl;
    }
    bool *input_list = new bool [il_size];
    for (long long int i =0; i < il_size; i++) {
        input_list[i] = true;
    }

    printf("Input List Size on CPU: %llu\n", il_size);



    // Pointers in GPU memory
    bool *dev_il;
    long long int *dev_pl;
    long long int *dev_input_size;
    long long int *dev_prime_size;
    long long int *dev_pl_end_number;   
    

    // Allocate the memory on the GPU
    gpuErrchk( hipMalloc( (void**)&dev_il,  il_size * sizeof(bool) ) );
    gpuErrchk( hipMalloc( (void**)&dev_pl,  small_sieve_counter * sizeof(long long int) ) );
    gpuErrchk( hipMalloc( (void**)&dev_input_size,  sizeof(long long int) ));
    gpuErrchk( hipMalloc( (void**)&dev_prime_size,  sizeof(long long int) ));
    gpuErrchk( hipMalloc( (void**)&dev_prime_size,  sizeof(long long int) ));
    gpuErrchk( hipMalloc( (void**)&dev_pl_end_number,  sizeof(long long int) ));


    // Copy the arrays 'a' and 'b' to the GPU
            gpuErrchk( hipMemcpy( dev_il, input_list, il_size * sizeof(bool),
             hipMemcpyHostToDevice ));
            gpuErrchk( hipMemcpy( dev_pl, prime_list, small_sieve_counter * sizeof(long long int),
             hipMemcpyHostToDevice ));
            gpuErrchk( hipMemcpy( dev_prime_size, &small_sieve_counter, sizeof(long long int),
             hipMemcpyHostToDevice ));
             gpuErrchk( hipMemcpy( dev_input_size, &il_size, sizeof(long long int),
             hipMemcpyHostToDevice ));
             gpuErrchk( hipMemcpy( dev_pl_end_number, &pl_end_number, sizeof(long long int),
             hipMemcpyHostToDevice ));


    //
    // GPU Calculation
    ////////////////////////

 //   printf("Running parallel job.\n");

    int grid_size = (small_sieve_counter-1)/block_size;
    grid_size++;

    if (DEBUG >=1) {
        cout << "Grid Size: " << grid_size << endl;
        cout << "Block Size: " << block_size << endl;
        
    }


    // ********************** KERNEL LAUNCH **********************
    gpuErrchk( hipProfilerStart() );

    if (DEBUG >=1) {
        cout << "Launching Kernel" << endl;
    }

    gpuErrchk(hipEventRecord(start,0));
    prime<<<grid_size,block_size>>>(dev_il, dev_pl, dev_input_size, dev_prime_size, dev_pl_end_number);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk(hipEventRecord(stop,0));
    gpuErrchk(hipEventSynchronize(stop));
    if (DEBUG >=1) {
        cout << "Kernel Computation Complete" << endl;
    }
    gpuErrchk(hipEventElapsedTime(&time, start, stop));
    yellow_start();
    printf("GPU Time: %.2f ms\n", time);
    color_reset();

        // Create Output list on CPU
        if (DEBUG >=1) {
            cout << "Allocating OUTPUT_LIST" << endl;
        }
        bool *output_list = new bool [il_size];
        

    // copy the array Input List back from the GPU to the CPU
    gpuErrchk(hipMemcpy( output_list, dev_il, il_size * sizeof(bool), 
             hipMemcpyDeviceToHost ));
    gpuErrchk(hipProfilerStop());

    // Check Returned Primes
    long long int ret_primes=0;
    
    for (long long int i = pl_end_number; i < pl_end_number*pl_end_number; i++) {
        if (output_list[i] == true) {
            // To display prime numbers
            //cout << i << " ";
            ret_primes++;
            //small_sieve_counter++;
        }
    }

    total_primes += ret_primes; 
    green_start();
    cout << "Total Primes: "<< total_primes;
    cout << endl;
    color_reset();
             
    

    // Free the memory allocated on the GPU
    hipFree( dev_il );
    hipFree( dev_pl );
    hipFree( dev_prime_size );
    hipFree( dev_input_size );
    hipFree( dev_pl_end_number );
    

     free(small_sieve);
     free(prime_list);
     free(input_list);
     free(output_list);



     cout << endl<< endl<< endl;
    return 0;
}

