#include "hip/hip_runtime.h"
#include <functions.h>
#include <debugger.h>

using namespace std;

// ********************** KERNEL DEFINITION **********************

__global__ void prime_generator(int* d_input_list, uint64_cu* d_prime_list, uint64_cu* d_startPrimelist,uint64_cu* d_total_inputsize,uint64_cu* d_number_of_primes)
{
 
    uint64_cu tid = (blockIdx.x*blockDim.x) + threadIdx.x;

     
        if (tid < d_number_of_primes[0]) {
            printf("Kaustubh\n");
                                uint64_cu primes=d_prime_list[tid];
                      //  printf("%llu\n",primes);
                        for(uint64_cu i=0;i<=d_total_inputsize[0];i++) // Added less than eual to here.
                        {
                                uint64_cu bucket= i/(WORD);
                                uint64_cu setbit= i%(WORD);
                                uint64_cu number=d_startPrimelist[0]+i;
                                
                        //      printf("%llu -----> hash the value %llu to %llu bucket and change the %llu bit\n",number,i,bucket,setbit );
                        //      printf("**************  %llu --- %llu \n",number,primes);
                                if(number%primes==0)
                                {
                                        printf("%llu is divisible by %llu \n", number,primes);
                                        d_input_list[bucket]=d_input_list[bucket]| 1U<<setbit;
                                }
                        }
        }
}



// ********************** PTHREAD ITERATION **********************

void *one_iteration(void *tid) {
    // Dont use tid
    // Use thread_id
    long gpu_id = (long) tid;



    if (DEBUG >= 1) {
        cout << "GPU Handler: " << gpu_id << endl;
    }

    hipEvent_t start, stop;
    

    

// Saurin's Code
kernelLauncher(gpu_id);



/*    
        // Check Returned Primes
        long long int ret_primes=0;
        
        for (long long int i = pl_end_number; i < pl_end_number*pl_end_number; i++) {
            if (output_list[i] == true) {
                // To display prime numbers
                //cout << i << " ";
                ret_primes++;
                //small_sieve_counter++;
            }
        }
    
        total_primes += ret_primes; 
        green_start();
        cout << "Total Primes: "<< total_primes;
        cout << endl;
        color_reset();
                 
        
    
        // Free the memory allocated on the GPU
        hipFree( dev_il );
        hipFree( dev_pl );
        hipFree( dev_prime_size );
        hipFree( dev_input_size );
        hipFree( dev_pl_end_number );
        
    
         free(small_sieve);
         free(prime_list);
         free(input_list);
         free(output_list);
    */
    

}




// Global Variables.
uint64_cu pl_end_number = 1000;
uint64_cu total_primes=0;
int number_of_gpus = 1;

PrimeHeader pheader;
GpuHandler gpu_data;
//long long int end_val = 1000000;


// ********************** MAIN FUNCTION **********************

int main(int argc, char *argv[]) { 

    start_info(); // Complete

    number_of_gpus = find_number_of_gpus(); // Complete
    number_of_gpus = pow(2,int(log(number_of_gpus)/log(2)));
    gpu_data.gpus = number_of_gpus;

    // Accepting input from Console
    switch (argc) { // For getting input from console
        case 6:
            //long input_5;
            //input_5 = atol(argv[5]); //Fifth Input
            
        case 5:
            //long input_4;
            //input_4 = atol(argv[4]); //Fourth Input
            
        case 4:
            //long input_3;
            //input_3 = atol(argv[3]); // Third Input
            
        case 3:
            long input_2;
            input_2 = atol(argv[2]); // Second Input
            number_of_gpus = (int)input_2; // Number of GPUs on the NODE.
            // Over-ride with input value.
        case 2:
            long input_1;
            input_1 = atol(argv[1]); // First input
            pl_end_number = (uint64_cu)input_1;

            break;
        case 1:
            // Keep this empty
            break;
        default:
            red_start();
            cout << "FATAL ERROR: Wrong Number of Inputs" << endl; // If incorrect number of inputs are used.
            color_reset();
            return 1;
    }


    




    //calculate_primes_on_cpu(); //TODO Store code in this function



    // Time Variables
    hipEvent_t start, stop;
    float time;
    gpuErrchk(hipEventCreate (&start));
    gpuErrchk(hipEventCreate (&stop));



    // Create Small 
    if (DEBUG >=2) {
        cout << "Allocating SMALL_SIEVE" << endl;
    }



    bool *small_sieve = new bool [pl_end_number];


    
    // Initialize Small Sieve
    for (uint64_cu i = 0; i < pl_end_number; i++) {
        small_sieve[i] = true;
    }

    // Compute Small Sieve on CPU
    hipEventRecord(start,0);
    
    for (uint64_cu i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (uint64_cu j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    gpuErrchk( hipEventRecord(stop,0));
    gpuErrchk( hipEventSynchronize(stop));
    gpuErrchk( hipEventElapsedTime(&time, start, stop));
    printf("CPU Time: %.2f ms\n", time);


    // Count Total Primes
    uint64_cu small_sieve_counter = 0;
    for (uint64_cu i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            // To display prime numbers
            //cout << i << " ";
            small_sieve_counter++;
        }
    }
    cout << endl;

    total_primes += small_sieve_counter;
    if (DEBUG >= 1) {
        cout << "Total Primes in Small Sieve: " << small_sieve_counter << endl;
    }



    if (DEBUG >=2) {
        cout << "Allocating PRIME_LIST" << endl;
    }


    uint64_cu *prime_list = new uint64_cu [small_sieve_counter];

    // Storing numbers from the sieve to an array.
    uint64_cu inner_counter = 0;
    for (uint64_cu i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }
    pheader.primelist=prime_list;
    pheader.length=small_sieve_counter;
    pheader.lastMaxNo=pl_end_number;

    



//    while(end_reached) {

    //  *************** Pthreads Launch *******************


    pthread_t *thread = new pthread_t [number_of_gpus];
    int *thread_error = new int [number_of_gpus];
    GpuHandler *handler = new GpuHandler [number_of_gpus];

    //initialize_handlers(handler);

    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_create(&thread[i], NULL, one_iteration, (void *) i);
        if (thread_error[i] && WARNINGS) {
            yellow_start();
            cout << "Warning: Thread " << i << " failed to launch" << endl;
            cout << "GPU: " << i << " is being mishandled." << endl;
            color_reset();
        }
    }
    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_join(thread[i], NULL);
    }

// output_combine();

    // INLINE
    //iteration_info();

//}


// CODE

    // INLINE
    //end_info();

    return 0;
}

