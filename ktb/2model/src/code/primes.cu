#include "hip/hip_runtime.h"
#include <functions.h>
#include <debugger.h>

using namespace std;


// Global Variables.
uint64_cu pl_end_number = 1000;
int number_of_gpus = 1;

PrimeHeader pheader;
GpuHandler gpu_data;
//long long int end_val = 1000000;




// ********************** KERNEL DEFINITION **********************

__global__ void prime_generator(int* d_input_list, uint64_cu* d_prime_list, uint64_cu* d_startPrimelist,uint64_cu* d_total_inputsize,uint64_cu* d_number_of_primes)
{
 
uint64_cu tid = (blockIdx.x*blockDim.x) + threadIdx.x;

     
if (tid < *d_number_of_primes) {
    //printf("Kaustubh\n");
    uint64_cu primes=d_prime_list[tid];
    for(uint64_cu i=0;i<=d_total_inputsize[0];i++) { // Added less than eual to here.
        uint64_cu bucket= i/(WORD);
        uint64_cu setbit= i%(WORD);
        uint64_cu number=d_startPrimelist[0]+i;
        if(number%primes==0) {
            //printf("%llu is divisible by %llu \n", number,primes);
            d_input_list[bucket]=d_input_list[bucket]| 1U<<setbit;
            }
        }
    }
}



// ********************** PTHREAD ITERATION **********************

void *one_iteration(void *tid) {
long gpu_id = (long) tid; // Dont use tid, Use gpu_id instead

if (DEBUG >= 1) {
    cout << "Launched GPU Handler: " << gpu_id << endl;
}

hipEvent_t start_kernel[gpu_data.gpus]; 
hipEvent_t stop_kernel[gpu_data.gpus];
float time[gpu_id];
gpuErrchk( hipEventCreate (&start_kernel[gpu_id]) );
gpuErrchk( hipEventCreate (&stop_kernel[gpu_id]) );

// hipStream_t stream[gpu_data.gpus];
// for (int i=0;i<gpu_data.gpus;i++) {
//     stream[i] = i;
// }

    


// Saurin's Code
gpu_data.IL_start = pl_end_number+1;
gpu_data.IL_end = pl_end_number*pl_end_number;

//gpuErrchk( hipEventRecord(start_kernel[gpu_id],(hipStream_t)gpu_id));
  
kernelLauncher(gpu_id);


//gpuErrchk( hipEventRecord(stop_kernel[gpu_id],(hipStream_t)gpu_id));
//gpuErrchk( hipEventSynchronize(stop_kernel[gpu_id]));
//gpuErrchk( hipEventElapsedTime(&time[gpu_id], start_kernel[gpu_id], stop_kernel[gpu_id]));
//printf("GPU %d Time: %.2f ms\n", gpu_id, time[gpu_id]);

}




// ********************** MAIN FUNCTION **********************

int main(int argc, char *argv[]) { 

    start_info(); // Complete

    number_of_gpus = find_number_of_gpus(); // Complete
    number_of_gpus = pow(2,int(log(number_of_gpus)/log(2)));
    gpu_data.gpus = number_of_gpus;

    // Accepting input from Console
    switch (argc) { // For getting input from console
        case 6:
            //long input_5;
            //input_5 = atol(argv[5]); //Fifth Input
            
        case 5:
            //long input_4;
            //input_4 = atol(argv[4]); //Fourth Input
            
        case 4:
            //long input_3;
            //input_3 = atol(argv[3]); // Third Input
            
        case 3:
            long input_2;
            input_2 = atol(argv[2]); // Second Input
            number_of_gpus = (int)input_2; // Number of GPUs on the NODE.
            // Over-ride with input value.
        case 2:
            long input_1;
            input_1 = atol(argv[1]); // First input
            pl_end_number = (uint64_cu)input_1;

            break;
        case 1:
            // Keep this empty
            break;
        default:
            red_start();
            cout << "FATAL ERROR: Wrong Number of Inputs" << endl; // If incorrect number of inputs are used.
            color_reset();
            return 1;
    }

    if (number_of_gpus != find_number_of_gpus()) {
        cyan_start();
        cout << "INFO: Running on " << number_of_gpus << " GPUs out of " << find_number_of_gpus() << " GPUs." << endl;
        color_reset();
    }

    pheader = calculate_primes_on_cpu(pheader,pl_end_number);
    
    cout << "pheader.length: " << pheader.length << endl;

//    while(end_reached) {

    //  *************** PTHREADS LAUNCH *******************


    pthread_t *thread = new pthread_t [number_of_gpus];
    int *thread_error = new int [number_of_gpus];

    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_create(&thread[i], NULL, one_iteration, (void *) i);
        if (thread_error[i] && WARNINGS) {
            yellow_start();
            cout << "Warning: Thread " << i << " failed to launch" << endl;
            cout << "GPU: " << i << " is being mishandled." << endl;
            color_reset();
        }
    }
    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_join(thread[i], NULL);
    }

// output_combine();

    // INLINE
    //iteration_info();

//}


// CODE

    // INLINE
    //end_info();

    return 0;
}

