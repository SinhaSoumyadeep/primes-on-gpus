#include "hip/hip_runtime.h"
#include <functions.h>
#include <debugger.h>

using namespace std;

#define block_size   32
#define DEBUG 1
#define GPU 0
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define WARNINGS 1


// ********************** KERNEL DEFINITION **********************

__global__ void prime( bool *il, 
    long long int *pl, 
    long long int *dev_input_size_ptr, 
    long long int *dev_prime_size_ptr, 
    long long int *dev_pl_end_number_ptr ) {
    
        long long int dev_input_size = *dev_input_size_ptr;
        long long int dev_prime_size = *dev_prime_size_ptr; 
        long long int dev_pl_end_number = *dev_pl_end_number_ptr;



        long long int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id


    if (tid <= dev_prime_size) {
        long long int tpno = pl[tid];
        //printf("\tTID: %d", tid);
            for (long long int k=dev_pl_end_number; k<dev_input_size; k++) {
                if (k % tpno == 0) {
                    il[k] = false;                   // add vectors together                
            }
        }
    }
}


// ********************** PTHREAD ITERATION **********************

void *one_iteration(void *tid) {
    // Dont use tid
    // Use thread_id
    long thread_id = (long) tid;

    if (DEBUG >= 1) {
        cout << "GPU Handler: " << thread_id << endl;
    }

    hipEvent_t start, stop;
    


        // Select GPU
        gpuErrchk(hipSetDevice(thread_id));


        // Pointers in GPU memory
        long long int *dev_prime_list;
        long long int *dev_prime_list_start;
        long long int *dev_prime_list_end;
        
        long long int *dev_input_list_start;
        long long int *dev_input_list_end;
        
        
    
        // Allocate the memory on the GPU
        gpuErrchk( hipMalloc( (void**)&dev_prime_list,  small_sieve_counter * sizeof(long long int) ) );
        gpuErrchk( hipMalloc( (void**)&dev_input_size,  sizeof(long long int) ));
        gpuErrchk( hipMalloc( (void**)&dev_prime_size,  sizeof(long long int) ));
        gpuErrchk( hipMalloc( (void**)&dev_prime_size,  sizeof(long long int) ));
        gpuErrchk( hipMalloc( (void**)&dev_pl_end_number,  sizeof(long long int) ));
    
    
        // Copy the arrays 'a' and 'b' to the GPU
                gpuErrchk( hipMemcpy( dev_il, input_list, il_size * sizeof(bool),
                 hipMemcpyHostToDevice ));
                gpuErrchk( hipMemcpy( dev_pl, prime_list, small_sieve_counter * sizeof(long long int),
                 hipMemcpyHostToDevice ));
                gpuErrchk( hipMemcpy( dev_prime_size, &small_sieve_counter, sizeof(long long int),
                 hipMemcpyHostToDevice ));
                 gpuErrchk( hipMemcpy( dev_input_size, &il_size, sizeof(long long int),
                 hipMemcpyHostToDevice ));
                 gpuErrchk( hipMemcpy( dev_pl_end_number, &pl_end_number, sizeof(long long int),
                 hipMemcpyHostToDevice ));
    
    
        //
        // GPU Calculation
        ////////////////////////
    
     //   printf("Running parallel job.\n");
    
        int grid_size = (small_sieve_counter-1)/block_size;
        grid_size++;
    
        if (DEBUG >=1) {
            cout << "Grid Size: " << grid_size << endl;
            cout << "Block Size: " << block_size << endl;
            
        }
    
    
        // ********************** KERNEL LAUNCH **********************
        gpuErrchk( hipProfilerStart() );
    
        if (DEBUG >=1) {
            cout << "Launching Kernel" << endl;
        }
    
        gpuErrchk(hipEventRecord(start,0));
        prime<<<grid_size,block_size>>>(dev_il, dev_pl, dev_input_size, dev_prime_size, dev_pl_end_number);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk(hipEventRecord(stop,0));
        gpuErrchk(hipEventSynchronize(stop));
        if (DEBUG >=2) {
            cout << "Kernel Computation Complete" << endl;
        }
        gpuErrchk(hipEventElapsedTime(&time, start, stop));
        yellow_start();
        printf("GPU Time: %.2f ms\n", time);
        color_reset();
    
            // Create Output list on CPU
            if (DEBUG >=2) {
                cout << "Allocating OUTPUT_LIST" << endl;
            }
            bool *output_list = new bool [il_size];
            
    
        // copy the array Input List back from the GPU to the CPU
        gpuErrchk(hipMemcpy( output_list, dev_il, il_size * sizeof(bool), 
                 hipMemcpyDeviceToHost ));
        gpuErrchk(hipProfilerStop());
    
        // Check Returned Primes
        long long int ret_primes=0;
        
        for (long long int i = pl_end_number; i < pl_end_number*pl_end_number; i++) {
            if (output_list[i] == true) {
                // To display prime numbers
                //cout << i << " ";
                ret_primes++;
                //small_sieve_counter++;
            }
        }
    
        total_primes += ret_primes; 
        green_start();
        cout << "Total Primes: "<< total_primes;
        cout << endl;
        color_reset();
                 
        
    
        // Free the memory allocated on the GPU
        hipFree( dev_il );
        hipFree( dev_pl );
        hipFree( dev_prime_size );
        hipFree( dev_input_size );
        hipFree( dev_pl_end_number );
        
    
         free(small_sieve);
         free(prime_list);
         free(input_list);
         free(output_list);
    
    

}




// Global Variables.
long long int pl_end_number = 1000;
long long int total_primes=0;
long number_of_gpus = 1;
//long long int end_val = 1000000;


// ********************** MAIN FUNCTION **********************

int main(int argc, char *argv[]) { 

    // INLINE
    start_info();

    number_of_gpus = find_number_of_gpus();


    // For debugging, I am hardcoding the numbber of GPUs to 1.
    number_of_gpus = 1; // Remove this line to run on multiple GPUs.


    // Accepting input from Console
    // INLINE
    console_input();


    







    // Time Variables
    hipEvent_t start, stop;
    float time;
    gpuErrchk(hipEventCreate (&start));
    gpuErrchk(hipEventCreate (&stop));



    // Create Small 
    if (DEBUG >=2) {
        cout << "Allocating SMALL_SIEVE" << endl;
    }



    bool *small_sieve = new bool [pl_end_number];


    
    // Initialize Small Sieve
    for (long long int i = 0; i < pl_end_number; i++) {
        small_sieve[i] = true;
    }

    // Compute Small Sieve on CPU
    hipEventRecord(start,0);
    
    for (long long int i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (long long int j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    gpuErrchk( hipEventRecord(stop,0));
    gpuErrchk( hipEventSynchronize(stop));
    gpuErrchk( hipEventElapsedTime(&time, start, stop));
    printf("CPU Time: %.2f ms\n", time);


    // Count Total Primes
    long long int small_sieve_counter = 0;
    for (long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            // To display prime numbers
            //cout << i << " ";
            small_sieve_counter++;
        }
    }
    cout << endl;

    total_primes += small_sieve_counter;
    if (DEBUG >= 1) {
        cout << "Total Primes in Small Sieve: " << small_sieve_counter << endl;
    }



    if (DEBUG >=2) {
        cout << "Allocating PRIME_LIST" << endl;
    }
    long long int *prime_list = new long long int [small_sieve_counter];

    // Storing numbers from the sieve to an array.
    long long int inner_counter = 0;
    for (long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }

    



    while(end_reached) {







    // Create Input list on CPU
    long long int il_size = pl_end_number*pl_end_number;
    if (DEBUG >=2) {
        cout << "Allocating INPUT_LIST" << endl;
    }
    bool *input_list = new bool [il_size];
    for (long long int i =0; i < il_size; i++) {
        input_list[i] = true;
    }

    if (DEBUG >=1) {
    printf("Input List Size on CPU: %llu\n", il_size);
    }



    // Pthreads Launch
    pthread_t *thread = new pthread_t [number_of_gpus];
    int *thread_error = new int [number_of_gpus];
    GpuHandler *handler = new GpuHandler [number_of_gpus];

    initialize_handlers(handler);

    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_create(&thread[i], NULL, one_iteration, (void *) i);
        if (thread_error[i] && WARNINGS) {
            yellow_start();
            cout << "Warning: Thread " << i << " failed to launch" << endl;
            cout << "GPU: " << i << " is being mishandled." << endl;
            color_reset();
        }
    }
    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_join(thread[i], NULL);
    }


    // INLINE
    iteration_info();

}


// CODE

    // INLINE
    end_info();

    return 0;
}

