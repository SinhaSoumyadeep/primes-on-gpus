#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <functions.h>
#include <debugger.h>

using namespace std;

#define block_size   32
#define vector_size 1000
#define DEBUG 1

__global__ void prime( bool *il, unsigned long long int *pl ) {
    int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

    if (tid <= sizeof(pl)/sizeof(unsigned long long int)) {
        unsigned long long int tpno = pl[tid];
            for (unsigned long long int k=0;k<sizeof(pl)/sizeof(bool);k++) {
                if (k % tpno == 0) {
                    il[k] = false;                   // add vectors together                
            }
        }
    }
}



// ********************** MAIN FUNCTION **********************

unsigned long long int pl_end_number = 1000;
//unsigned long long int end_val = 1000000;


int main(int argc, char *argv[]) { 

    

    green_start();
    cout << "\n\n\n\n\n\n\n\n\n\nProgram Start\n";
    color_reset();
    
    // Accepting input from Console
    switch (argc) { // For getting input from console
        case 6:
            long input_5;
            input_5 = atol(argv[5]); //Fifth Input
            //num_threads = input_5;
        case 5:
            long input_4;
            input_4 = atol(argv[4]); //Fourth Input
            //in_parallel = input_4;
        case 4:
            long input_3;
            input_3 = atol(argv[3]); // Third Input
            //display_bit = input_3;
        case 3:
            long input_2;
            input_2 = atol(argv[2]); // Second Input
        case 2:
            long input_1;
            input_1 = atol(argv[1]); // First input
            pl_end_number = (unsigned long long int)input_1;

            break;
        case 1:
            // Keep this empty
            break;
        default:
            red_start();
            cout << "FATAL ERROR: Wrong Number of Inputs" << endl; // If incorrect number of inputs are used.
            color_reset();
            return 1;
    }








    // Select GPU
    hipSetDevice(1);

    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);


    // Create Small Sieve
    bool *small_sieve = new bool [pl_end_number];


    
    // Initialize Small Sieve
    for (unsigned long long int i = 0; i < pl_end_number; i++) {
        small_sieve[i] = true;
    }

    // Compute Small Sieve on CPU
    hipEventRecord(start,0);
    
    for (unsigned long long int i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (unsigned long long int j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("CPU Time: %.2f ms\n", time);


    // Count Total Primes
    unsigned long long int small_sieve_counter = 0;
    for (unsigned long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            // To display prime numbers
            //cout << i << " ";
            small_sieve_counter++;
        }
    }
    cout << endl;


    unsigned long long int *prime_list = new unsigned long long int [small_sieve_counter];

    // Storing numbers from the sieve to an array.
    unsigned long long int inner_counter = 0;
    for (unsigned long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }

    
    // Create Input list on CPU
    unsigned long long int il_size = pl_end_number*pl_end_number;
    bool *input_list = new bool [il_size];
    for (unsigned long long int i =0; i < il_size; i++) {
        input_list[i] = true;
    }

    // Pointers in GPU memory
    bool *dev_il;
    unsigned long long int *dev_pl;
    

    // Allocate the memory on the GPU
    hipMalloc( (void**)&dev_il,  il_size * sizeof(bool) );
    hipMalloc( (void**)&dev_pl,  small_sieve_counter * sizeof(unsigned long long int) );


    // Copy the arrays 'a' and 'b' to the GPU
     hipMemcpy( dev_il, input_list, il_size * sizeof(bool),
             hipMemcpyHostToDevice );
     hipMemcpy( dev_pl, prime_list, small_sieve_counter * sizeof(unsigned long long int),
             hipMemcpyHostToDevice );


    //
    // GPU Calculation
    ////////////////////////

 //   printf("Running parallel job.\n");

    int grid_size = (small_sieve_counter-1)/block_size;
    grid_size++;

    hipEventRecord(start,0);
    prime<<<grid_size,block_size>>>(dev_il, dev_pl);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    printf("GPU Time: %.2f ms\n", time);

        // Create Output list on CPU
        bool *output_list = new bool [il_size];
        

    // copy the array Input List back from the GPU to the CPU
     hipMemcpy( output_list, dev_il, il_size * sizeof(bool), 
             hipMemcpyDeviceToHost );


    // Check Returned Primes
    for (unsigned long long int i = pl_end_number; i < pl_end_number*pl_end_number; i++) {
        if (output_list[i] == true) {
            // To display prime numbers
            cout << i << " ";
            //small_sieve_counter++;
        }
    }
    cout << endl;
             
    

    // Free the memory allocated on the GPU
    hipFree( dev_il );
    hipFree( dev_pl );

    // free(a);
    // free(b);
    // free(c_cpu);
    // free(c_gpu);

    return 0;
}

