#include "hip/hip_runtime.h"
#include <functions.h>
#include <debugger.h>

using namespace std;

#define block_size   32
#define DEBUG 1
#define GPU 0
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define WARNINGS 1


// ********************** KERNEL DEFINITION **********************

__global__ void prime_generator(int* d_input_list, uint64_cu* d_prime_list, uint64_cu* d_startPrimelist,uint64_cu* d_total_inputsize,uint64_cu* d_number_of_primes)
{
        long long int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
        if (tid < d_number_of_primes[0]) {
                                uint64_cu primes=d_prime_list[tid];
                      //  printf("%llu\n",primes);
                        for(uint64_cu i=0;i<=d_total_inputsize[0];i++) // Added less than eual to here.
                        {
                                uint64_cu bucket= i/(WORD);
                                uint64_cu setbit= i%(WORD);
                                uint64_cu number=d_startPrimelist[0]+i;
                        //      printf("%llu -----> hash the value %llu to %llu bucket and change the %llu bit\n",number,i,bucket,setbit );
                        //      printf("**************  %llu --- %llu \n",number,primes);
                                if(number%primes==0)
                                {
                                        d_input_list[bucket]=d_input_list[bucket]| 1U<<setbit;
                                }
                        }
        }
}



// ********************** PTHREAD ITERATION **********************

void *one_iteration(void *tid) {
    // Dont use tid
    // Use thread_id
    long gpu_id = (long) tid;



    if (DEBUG >= 1) {
        cout << "GPU Handler: " << thread_id << endl;
    }

    hipEvent_t start, stop;
    


// Saurin's Code



        // Select GPU
        gpuErrchk(hipSetDevice(thread_id));


        // Pointers in GPU memory
        long long int *dev_prime_list;
        long long int *dev_prime_list_start;
        long long int *dev_prime_list_end;
        
        long long int *dev_input_list_start;
        long long int *dev_input_list_end;
        
        
    
        // Allocate the memory on the GPU
        gpuErrchk( hipMalloc( (void**)&dev_prime_list,  small_sieve_counter * sizeof(long long int) ) );
        gpuErrchk( hipMalloc( (void**)&dev_input_size,  sizeof(long long int) ));
        gpuErrchk( hipMalloc( (void**)&dev_prime_size,  sizeof(long long int) ));
        gpuErrchk( hipMalloc( (void**)&dev_prime_size,  sizeof(long long int) ));
        gpuErrchk( hipMalloc( (void**)&dev_pl_end_number,  sizeof(long long int) ));
    
    
        // Copy the arrays 'a' and 'b' to the GPU
                gpuErrchk( hipMemcpy( dev_il, input_list, il_size * sizeof(bool),
                 hipMemcpyHostToDevice ));
                gpuErrchk( hipMemcpy( dev_pl, prime_list, small_sieve_counter * sizeof(long long int),
                 hipMemcpyHostToDevice ));
                gpuErrchk( hipMemcpy( dev_prime_size, &small_sieve_counter, sizeof(long long int),
                 hipMemcpyHostToDevice ));
                 gpuErrchk( hipMemcpy( dev_input_size, &il_size, sizeof(long long int),
                 hipMemcpyHostToDevice ));
                 gpuErrchk( hipMemcpy( dev_pl_end_number, &pl_end_number, sizeof(long long int),
                 hipMemcpyHostToDevice ));
    
    
        //
        // GPU Calculation
        ////////////////////////
    
     //   printf("Running parallel job.\n");
    
        int grid_size = (small_sieve_counter-1)/block_size;
        grid_size++;
    
        if (DEBUG >=1) {
            cout << "Grid Size: " << grid_size << endl;
            cout << "Block Size: " << block_size << endl;
            
        }
    
    
        // ********************** KERNEL LAUNCH **********************
        gpuErrchk( hipProfilerStart() );
    
        if (DEBUG >=1) {
            cout << "Launching Kernel" << endl;
        }
    
        gpuErrchk(hipEventRecord(start,0));
        prime<<<grid_size,block_size>>>(dev_il, dev_pl, dev_input_size, dev_prime_size, dev_pl_end_number);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk(hipEventRecord(stop,0));
        gpuErrchk(hipEventSynchronize(stop));
        if (DEBUG >=2) {
            cout << "Kernel Computation Complete" << endl;
        }
        gpuErrchk(hipEventElapsedTime(&time, start, stop));
        yellow_start();
        printf("GPU Time: %.2f ms\n", time);
        color_reset();
    
            // Create Output list on CPU
            if (DEBUG >=2) {
                cout << "Allocating OUTPUT_LIST" << endl;
            }
            bool *output_list = new bool [il_size];
            
    
        // copy the array Input List back from the GPU to the CPU
        gpuErrchk(hipMemcpy( output_list, dev_il, il_size * sizeof(bool), 
                 hipMemcpyDeviceToHost ));
        gpuErrchk(hipProfilerStop());
    
        // Check Returned Primes
        long long int ret_primes=0;
        
        for (long long int i = pl_end_number; i < pl_end_number*pl_end_number; i++) {
            if (output_list[i] == true) {
                // To display prime numbers
                //cout << i << " ";
                ret_primes++;
                //small_sieve_counter++;
            }
        }
    
        total_primes += ret_primes; 
        green_start();
        cout << "Total Primes: "<< total_primes;
        cout << endl;
        color_reset();
                 
        
    
        // Free the memory allocated on the GPU
        hipFree( dev_il );
        hipFree( dev_pl );
        hipFree( dev_prime_size );
        hipFree( dev_input_size );
        hipFree( dev_pl_end_number );
        
    
         free(small_sieve);
         free(prime_list);
         free(input_list);
         free(output_list);
    
    

}




// Global Variables.
long long int pl_end_number = 1000;
long long int total_primes=0;
long number_of_gpus = 1;
PrimeHeader pheader;
GpuHandler gpu_data;
//long long int end_val = 1000000;


// ********************** MAIN FUNCTION **********************

int main(int argc, char *argv[]) { 

    // INLINE
    start_info(); // Complete

    number_of_gpus = find_number_of_gpus(); // Complete
    number_of_gpus = pow(2,int(log(number_of_gpus)/log(2)));
    gpu_data->gpus = number_of_gpus;

    // Accepting input from Console
    // INLINE
    console_input(); // Complete


    




    //calculate_primes_on_cpu(); //TODO Store code in this function



    // Time Variables
    hipEvent_t start, stop;
    float time;
    gpuErrchk(hipEventCreate (&start));
    gpuErrchk(hipEventCreate (&stop));



    // Create Small 
    if (DEBUG >=2) {
        cout << "Allocating SMALL_SIEVE" << endl;
    }



    bool *small_sieve = new bool [pl_end_number];


    
    // Initialize Small Sieve
    for (long long int i = 0; i < pl_end_number; i++) {
        small_sieve[i] = true;
    }

    // Compute Small Sieve on CPU
    hipEventRecord(start,0);
    
    for (long long int i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (long long int j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                //cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    gpuErrchk( hipEventRecord(stop,0));
    gpuErrchk( hipEventSynchronize(stop));
    gpuErrchk( hipEventElapsedTime(&time, start, stop));
    printf("CPU Time: %.2f ms\n", time);


    // Count Total Primes
    long long int small_sieve_counter = 0;
    for (long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            // To display prime numbers
            //cout << i << " ";
            small_sieve_counter++;
        }
    }
    cout << endl;

    total_primes += small_sieve_counter;
    if (DEBUG >= 1) {
        cout << "Total Primes in Small Sieve: " << small_sieve_counter << endl;
    }



    if (DEBUG >=2) {
        cout << "Allocating PRIME_LIST" << endl;
    }


    long long int *prime_list = new long long int [small_sieve_counter];

    // Storing numbers from the sieve to an array.
    long long int inner_counter = 0;
    for (long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }
    pheader.primelist=prime_list;
    pheader.length=small_sieve_counter;
    pheader.lastMaxNo=pl_end_number;

    



//    while(end_reached) {

    //  *************** Pthreads Launch *******************


    pthread_t *thread = new pthread_t [number_of_gpus];
    int *thread_error = new int [number_of_gpus];
    GpuHandler *handler = new GpuHandler [number_of_gpus];

    initialize_handlers(handler);

    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_create(&thread[i], NULL, one_iteration, (void *) i);
        if (thread_error[i] && WARNINGS) {
            yellow_start();
            cout << "Warning: Thread " << i << " failed to launch" << endl;
            cout << "GPU: " << i << " is being mishandled." << endl;
            color_reset();
        }
    }
    for (long i = 0; i < number_of_gpus; i++) {
        thread_error[i] = pthread_join(thread[i], NULL);
    }

// output_combine();

    // INLINE
    //iteration_info();

//}


// CODE

    // INLINE
    //end_info();

    return 0;
}

