
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

using namespace std;

#define block_size   32
#define pl_end_number 100
#define vector_size 1000

__global__ void prime( int *a, int *b, int *c ) {
    int tid = (blockIdx.x*blockDim.x) + threadIdx.x;    // this thread handles the data at its thread id

    if (tid < vector_size){
        c[tid] = a[tid] + b[tid];                   // add vectors together                
    }
}



// ********************** MAIN FUNCTION **********************


int main( void ) { 

    cout << "Program Start" << endl;

    hipSetDevice(0);

    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    bool *small_sieve = new bool [pl_end_number];

    for (unsigned long long int i = 0; i < pl_end_number; i++) {
        small_sieve[i] = true;
    }

    for (unsigned long long int i = 2; i <= int(sqrt(pl_end_number))+1; i++) {
        for (unsigned long long int j = i+1; j <= pl_end_number; j++) {
            if (j % i == 0) {
                small_sieve[j] = false;
                cout << j << " is Composite, as divisible by " << i << endl;
            }
        }        
    }

    cout << "Primes till 100\n";

    unsigned long long int small_sieve_counter = 0;
    for (unsigned long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            cout << i << " ";
            small_sieve_counter++;
            //cout << small_sieve[i] << "    ";
        }
    }
    cout << endl;

    unsigned long long int *prime_list = new unsigned long long int [small_sieve_counter];

    unsigned long long int inner_counter = 0;
    for (unsigned long long int i = 2; i <= pl_end_number; i++) {
        if (small_sieve[i] == true) {
            prime_list[inner_counter] = i;
            inner_counter++;
        }
    }

    


    // Input Arrays and variables
    int *input_list        = new int [vector_size]; 
    int *c_cpu    = new int [vector_size]; 
    int *c_gpu    = new int [vector_size];

    // Pointers in GPU memory
    int *dev_a;
    int *dev_b;
    int *dev_c;



    //
    // CPU Calculation
    //////////////////

  //  printf("Running sequential job.\n");
    hipEventRecord(start,0);

    // Calculate C in the CPU
    // for (int i = 0; i < vector_size; i++) {
    //     c_cpu[i] = a[i] + b[i];
    // }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
 //   printf("\tSequential Job Time: %.2f ms\n", time);

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a,  vector_size * sizeof(int) );
    hipMalloc( (void**)&dev_b,  vector_size * sizeof(int) );
    hipMalloc( (void**)&dev_c,  vector_size * sizeof(int) );

    // copy the arrays 'a' and 'b' to the GPU
    // cudaMemcpy( dev_a, a, vector_size * sizeof(int),
    //         cudaMemcpyHostToDevice );
    // cudaMemcpy( dev_b, b, vector_size * sizeof(int),
    //         cudaMemcpyHostToDevice );


    //
    // GPU Calculation
    ////////////////////////

 //   printf("Running parallel job.\n");

    int grid_size = (vector_size-1)/block_size;
    grid_size++;

    hipEventRecord(start,0);
    //prime<<<grid_size,block_size>>>( dev_a, dev_b, dev_c);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
 //   printf("\tParallel Job Time: %.2f ms\n", time);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy( c_gpu, dev_c, vector_size * sizeof(int), 
            hipMemcpyDeviceToHost );

    // compare the results
    int error = 0;
    for (int i = 0; i < vector_size; i++) {
        if (c_cpu[i] != c_gpu[i]){
            error = 1;
            // printf( "Error starting element %d, %d != %d\n", i, c_gpu[i], c_cpu[i] );    
        }
        if (error) break; 
    }

    // if (error == 0){
    //     printf ("Correct result. No errors were found.\n");
    // }

    // free the memory allocated on the GPU
    // cudaFree( dev_a );
    // cudaFree( dev_b );
    // cudaFree( dev_c );

    // free(a);
    // free(b);
    // free(c_cpu);
    // free(c_gpu);

    return 0;
}

