#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define EXECCPU 0

#define block_size   32
#define vector_size  1000000000

#define LIMIT 10

#define ROWS 3000
#define K 4000
#define COLS 5000
#define INTSIZE sizeof(unsigned int)
#define BLOCK_SIZE 32 

void printList(int* ilist, int len){
    printf("\n(START, length-> %d)\n", len);
    for(int index=0; index<len ; index++){
        printf(" %d",ilist[index]);
    }
    printf("\nEND \n");
}

int countPrime(int* arr, int len){
    int pcount = 0;
    for(int x=0; x<len; x++){
        if(arr[x]!=-1)pcount++;
    }
    return pcount;
}

void addPrimes(int* target, int* source, int sourcelen){
    int pindex = 0;
    for(int val=0; val<sourcelen; val++){
        if(source[val]!=-1){
            target[pindex] = source[val];
            pindex++;
        }
    }
}

__global__ void calcPrime(int* primelist, int* inputlist,unsigned  int plen, unsigned int ilen ){

    unsigned int ind1 = blockIdx.x * blockDim.x + threadIdx.x;
    int num = primelist[ind1];
    int lastno = inputlist[ilen-1];

    //printf("%d --- %d \n",num, lastno);

    if(num<lastno){
        for(int start = 0; start< ilen; start++){
            if(inputlist[start] % num == 0){
                inputlist[start] = -1;
            }
        }
    }
}

int main( void ) { 

    // Set device that we will use for our cuda code
    // It will be either 0 or 1
    hipSetDevice(1);
    srand(time(NULL));
    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    int firstLimit = LIMIT;
    printf("firstLimit %d \n", firstLimit);

    int* firstLimitArray;
    int firstLimitLen = firstLimit-1;
    printf("firstLimitLen %d \n", firstLimitLen);
    hipHostMalloc( (void**)&firstLimitArray,  firstLimitLen*INTSIZE);

    for(int x=2; x<= firstLimit; x++){
        //printf(" %d %d \t",x-2,x);
        firstLimitArray[x-2] = x;
    }
    //printList(firstLimitArray, firstLimitLen);

    hipEventRecord(start,0);

    for(int val = 0; val < firstLimitLen/2; val++){
        int num = firstLimitArray[val];
        if(num==-1) continue;
        //printf("\n fixing prime %d ", num);
        for(int index=val+1; index< firstLimitLen; index++){
            //printf(" %d, %d ", num, firstLimitArray[index]);
            if(firstLimitArray[index]%num== 0 && firstLimitArray[index]!=0)
                firstLimitArray[index] = -1;
        }
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\nSerial Job Time: %.2f ms\n", time);

    //printList(firstLimitArray, firstLimitLen);
    int pcount = countPrime(firstLimitArray, firstLimitLen);
    printf("first round primes %d",pcount);

    int* primelist ;
    int plen = pcount;
    hipHostMalloc( (void**)&primelist,  pcount*INTSIZE);

    addPrimes(primelist, firstLimitArray, firstLimitLen);
    /*
    int pindex = 0;
    for(int val=0; val<firstLimitLen; val++){
        if(firstLimitArray[val]!=-1){
            primelist[pindex] = firstLimitArray[val];
            pindex++;
        }
    }
    printf("\n pindex -> %d", pindex);
    */

    int CUR_MAX = firstLimit;

    int startNo = CUR_MAX+1;
    int endNo = CUR_MAX * CUR_MAX; 

    int range = endNo - CUR_MAX;
    printf(" \n range %d",range);
    int* inputlist ;
    hipHostMalloc((void**)&inputlist,range*INTSIZE);

    for(int index = 0; index < range; index++){
        inputlist[index] = index + startNo;
    }

    //printList(inputlist,range);

    // Pointers in GPU memory
    int *dev_ilist;
    int *dev_plist;

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_plist,  plen*INTSIZE);
    hipMalloc( (void**)&dev_ilist,  range*INTSIZE);

    hipMemcpy( dev_plist, primelist, plen*INTSIZE, hipMemcpyHostToDevice );
    hipMemcpy( dev_ilist, inputlist, range*INTSIZE, hipMemcpyHostToDevice );

    //
    // GPU Calculation
    ////////////////////////
    unsigned int gridSize =  (plen + BLOCK_SIZE - 1)/ BLOCK_SIZE; 
    dim3 grids(gridSize,1);
    dim3 blocks(BLOCK_SIZE, 1);

    hipEventRecord(start,0);
    calcPrime<<<grids, blocks>>>(dev_plist, dev_ilist, plen, range);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy( inputlist, dev_ilist, range*INTSIZE, hipMemcpyHostToDevice );
    printf("\nParallel Job Time: %.2f ms\n", time);
    printList(inputlist,range);
    return 0;
}
