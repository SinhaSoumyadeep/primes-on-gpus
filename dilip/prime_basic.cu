#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "primedisk.h"

#define EXECCPU 0

#define LIMIT 100

typedef unsigned long long int uint64_cu;
#define INTSIZE sizeof(uint64_cu)
#define BLOCK_SIZE 32 

uint64_cu countPrime(uint64_cu* arr, uint64_cu len){
    uint64_cu pcount = 0;
    for(uint64_cu x=0; x<len; x++){
        if(arr[x]!=0)pcount++;
    }
    return pcount;
}

void addPrimes(uint64_cu* target, uint64_cu* source,uint64_cu sourcelen){
    uint64_cu pindex = 0;
    for(uint64_cu val=0; val<sourcelen; val++){
        if(source[val]!=0){
            target[pindex] = source[val];
            pindex++;
        }
    }
}

__global__ void calcPrime(uint64_cu* primelist, uint64_cu* inputlist,uint64_cu plen, uint64_cu ilen ){

    uint64_cu ind1 = blockIdx.x * blockDim.x + threadIdx.x;
    //uint64_cu num = primelist[ind1-1];
    //uint64_cu lastno = inputlist[ilen-1];

    /*
       if(num > 99403){
       printf("calcPrime %lu --- %lu \n",num, lastno);
       }
     */

    //printf("\n threadId %llu , ilen %llu, plen %llu",ind1,ilen, plen);

    if(ind1<plen){
        uint64_cu num = primelist[ind1];
        //printf("\ncore num %llu\n",num);
        //uint64_cu lastno = inputlist[ilen-1];
        for(uint64_cu start = 0; start< ilen; start++){
            if(inputlist[start] == num) continue;
            if(inputlist[start] % num == 0){
                //printf("CROSSING %llu --- %llu \n",num, inputlist[start]);
                inputlist[start] = 0;
            }
        }
    }
}

int main( void ) { 
    hipSetDevice(0);
    srand(time(NULL));
    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);


    // Set device that we will use for our cuda code
    // It will be either 0 or 1

    PrimeHeader ret = readPrimes();
    uint64_cu firstLimit = ret.lastMaxNo;
    uint64_cu plen = ret.length;
    uint64_cu* primelist = ret.primelist;
    printf("\n\n ret lastMaxNo-> %llu ",ret.lastMaxNo);
    printf("\tlength -> %llu ",ret.length);
    printList(ret.primelist ,ret.length);

    printf(" \n\n>>>>>>>>>>>>>> POST FILE READ\n");
    if(ret.length == 0 ){
        // start from beginning on CPU
        firstLimit = 10;
        printf("firstLimit %llu \n", firstLimit);

        uint64_cu firstLimitLen = firstLimit-1;
        printf("firstLimitLen %llu \n", firstLimitLen);
        uint64_cu* firstLimitArray = (uint64_cu*) malloc(firstLimitLen*INTSIZE);

        for(uint64_cu x=2; x<= firstLimit; x++){
            //printf(" %d %d \t",x-2,x);
            firstLimitArray[x-2] = x;
        }
        //printList(firstLimitArray, firstLimitLen);

        hipEventRecord(start,0);

        for(uint64_cu val = 0; val < firstLimitLen/2; val++){
            uint64_cu num = firstLimitArray[val];
            if(num==0) continue;
            //printf("\n fixing prime %llu ", num);
            for(uint64_cu index=val+1; index< firstLimitLen; index++){
                //printf(" %llu, %llu ", num, firstLimitArray[index]);
                if(firstLimitArray[index]%num== 0 && firstLimitArray[index]!=0)
                    firstLimitArray[index] = 0;
            }
        }
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        //printList(firstLimitArray, firstLimitLen);
        printf("\nSerial Job Time: %.2f ms\n", time);
        //printList(firstLimitArray, firstLimitLen);
        uint64_cu pcount = countPrime(firstLimitArray, firstLimitLen);
        //printf("first round primes %llu",pcount);

        plen = pcount;
        primelist = (uint64_cu*) malloc(pcount*INTSIZE);
        addPrimes(primelist, firstLimitArray, firstLimitLen);
        writePrimes(primelist,plen,firstLimit);
    } 

    while(firstLimit < LIMIT*LIMIT){
        printf("\nfirstLimit %llu",firstLimit);
        uint64_cu CUR_MAX = firstLimit;

        uint64_cu startNo = CUR_MAX+1;
        uint64_cu endNo = CUR_MAX * CUR_MAX; 

        uint64_cu range = endNo - CUR_MAX;
        printf("\n######################## startNo %llu , endNo %llu  ########################", startNo, endNo);
        //printf("\nrange %llu\n",range);
        uint64_cu* inputlist = (uint64_cu*) malloc(range*INTSIZE);

        for(uint64_cu index = 0; index < range; index++){
            inputlist[index] = index + startNo;
        }

        //printList(inputlist,range);

        // Pointers in GPU memory
        uint64_cu* dev_ilist;
        uint64_cu* dev_plist;

        // allocate the memory on the GPU
        hipMalloc( (void**)&dev_plist,  plen*INTSIZE);
        hipMalloc( (void**)&dev_ilist,  range*INTSIZE);

        hipMemcpy( dev_plist, primelist, plen*INTSIZE, hipMemcpyHostToDevice );
        hipMemcpy( dev_ilist, inputlist, range*INTSIZE, hipMemcpyHostToDevice );

        //
        // GPU Calculation
        ////////////////////////
        uint64_cu gridSize =  ((plen + BLOCK_SIZE - 1)/ BLOCK_SIZE) + 1; 
        hipEventRecord(start,0);
        calcPrime<<<gridSize, BLOCK_SIZE>>>(dev_plist, dev_ilist, plen, range);

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        hipMemcpy( inputlist, dev_ilist, range*INTSIZE, hipMemcpyDeviceToHost);
        printf("\n\nUpto %llu , Parallel Job Time: %.2f ms\n",endNo ,time);
        //printList(inputlist,range);

        // 2) WRITE primes from INPUTLIST
        uint64_cu ilistPrimeCount = countPrime(inputlist,range);
        printf("ilistPrimeCount %llu \n",ilistPrimeCount);
        uint64_cu* ilistprimes = (uint64_cu*) malloc(ilistPrimeCount*INTSIZE);
        addPrimes(ilistprimes, inputlist, range);
        writePrimes(ilistprimes,ilistPrimeCount,endNo);
        //printList(ilistprimes,ilistPrimeCount);

        // APPEND LOGIC
        uint64_cu totalPrimes = plen + ilistPrimeCount;
        printf("\n%llu totalPrimes for Upto %llu",totalPrimes,endNo);
        uint64_cu* primeNewArray = (uint64_cu*) malloc(totalPrimes*INTSIZE);
        memcpy(primeNewArray,primelist,plen*INTSIZE);
        memcpy(primeNewArray+plen, ilistprimes, ilistPrimeCount*INTSIZE);
        //printList(primeNewArray, totalPrimes);

        primelist = primeNewArray;
        plen = totalPrimes;
        firstLimit = endNo;
        fflush(stdout);
    }

    printf("\n**** MAIN END ***\n");
    return 0;
}
