#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define EXECCPU 0

#define block_size   32
#define vector_size  1000000000

#define LIMIT 10

#define ROWS 3000
#define K 4000
#define COLS 5000
#define INTSIZE sizeof(unsigned int)
#define BLOCK_SIZE 32 

void printList(int* ilist, int len){
    printf("\n(START, length-> %d)\n", len);
    for(int index=0; index<len ; index++){
        printf("%d ",ilist[index]);
    }
    printf("\nEND \n");
}

int countPrime(int* arr, int len){
    int pcount = 0;
    for(int x=0; x<len; x++){
        if(arr[x]!=-1)pcount++;
    }
    return pcount;
}

void addPrimes(int* target, int* source, int sourcelen){
    int pindex = 0;
    for(int val=0; val<sourcelen; val++){
        if(source[val]!=-1){
            target[pindex] = source[val];
            pindex++;
        }
    }
}

__global__ void calcPrime(int* primelist, int* inputlist,unsigned  int plen, unsigned int ilen ){

    unsigned int ind1 = blockIdx.x * blockDim.x + threadIdx.x;
    int num = primelist[ind1];
    int lastno = inputlist[ilen-1];

    //printf("%d --- %d \n",num, lastno);

    if(num<lastno){
        for(int start = 0; start< ilen; start++){
            if(inputlist[start] % num == 0){
                //printf("CROSSING %d --- %d \n",num, inputlist[start]);
                inputlist[start] = -1;
            }
        }
    }
}

int main( void ) { 

    // Set device that we will use for our cuda code
    // It will be either 0 or 1
    hipSetDevice(1);
    srand(time(NULL));
    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    int firstLimit = LIMIT;
    printf("firstLimit %d \n", firstLimit);

    int firstLimitLen = firstLimit-1;
    printf("firstLimitLen %d \n", firstLimitLen);
    int* firstLimitArray = (int*) malloc(firstLimitLen*INTSIZE);

    for(int x=2; x<= firstLimit; x++){
        //printf(" %d %d \t",x-2,x);
        firstLimitArray[x-2] = x;
    }
    //printList(firstLimitArray, firstLimitLen);

    hipEventRecord(start,0);

    for(int val = 0; val < firstLimitLen/2; val++){
        int num = firstLimitArray[val];
        if(num==-1) continue;
        //printf("\n fixing prime %d ", num);
        for(int index=val+1; index< firstLimitLen; index++){
            //printf(" %d, %d ", num, firstLimitArray[index]);
            if(firstLimitArray[index]%num== 0 && firstLimitArray[index]!=0)
                firstLimitArray[index] = -1;
        }
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    //printList(firstLimitArray, firstLimitLen);
    printf("\nSerial Job Time: %.2f ms\n", time);

    //printList(firstLimitArray, firstLimitLen);
    int pcount = countPrime(firstLimitArray, firstLimitLen);
    printf("first round primes %d",pcount);

    int plen = pcount;
    int* primelist = (int*) malloc(pcount*INTSIZE);

    addPrimes(primelist, firstLimitArray, firstLimitLen);

    int CUR_MAX = firstLimit;

    int startNo = CUR_MAX+1;
    int endNo = CUR_MAX * CUR_MAX; 

    int range = endNo - CUR_MAX;
    printf("\n range %d",range);
    int* inputlist = (int*) malloc(range*INTSIZE);

    for(int index = 0; index < range; index++){
        inputlist[index] = index + startNo;
    }

    //printList(inputlist,range);

    // Pointers in GPU memory
    int *dev_ilist;
    int *dev_plist;

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_plist,  plen*INTSIZE);
    hipMalloc( (void**)&dev_ilist,  range*INTSIZE);

    hipMemcpy( dev_plist, primelist, plen*INTSIZE, hipMemcpyHostToDevice );
    hipMemcpy( dev_ilist, inputlist, range*INTSIZE, hipMemcpyHostToDevice );

    //
    // GPU Calculation
    ////////////////////////
    unsigned int gridSize =  ((plen + BLOCK_SIZE - 1)/ BLOCK_SIZE) + 1; 
    hipEventRecord(start,0);
    calcPrime<<<gridSize, BLOCK_SIZE>>>(dev_plist, dev_ilist, plen, range);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy( inputlist, dev_ilist, range*INTSIZE, hipMemcpyDeviceToHost);
    printf("\nParallel Job Time: %.2f ms\n", time);
    //printList(inputlist,range);

    printf("\n plen %d ",plen);
    FILE* fout = fopen("pdata.txt","w");
    //fprintf(fout,"%d",plen);
    fwrite(&plen, INTSIZE, 1, fout);
    fwrite(primelist, INTSIZE, plen, fout );
    printList(primelist,plen);

    int ilistPrimeCount = countPrime(inputlist,range);
    printf("ilistPrimeCount %d",ilistPrimeCount);
    int* ilistprimes = (int*) malloc(ilistPrimeCount*INTSIZE);

    addPrimes(ilistprimes, inputlist, range);
    //fprintf(fout,"%d",ilistPrimeCount);
    fwrite(&ilistPrimeCount, INTSIZE, 1, fout);
    fwrite(ilistprimes, INTSIZE, ilistPrimeCount, fout );
    printList(ilistprimes,ilistPrimeCount);

    // APPEND LOGIC
    int totalPrimes = plen + ilistPrimeCount;
    printf("\n%d totalPrimes ",totalPrimes);
    int* primeNewArray = (int*) malloc(totalPrimes*INTSIZE);
    memcpy(primeNewArray,primelist,plen*INTSIZE);
    memcpy(primeNewArray+plen, ilistprimes, ilistPrimeCount*INTSIZE);
    printList(primeNewArray, totalPrimes);

    /* this caused segfault but above one does not
    int* primeListNow = (int*)realloc(primelist, totalPrimes);
    printList(primeListNow, totalPrimes);
    //memcpy(primeListNow+plen, ilistprimes,ilistPrimeCount*INTSIZE);
    memcpy(primeListNow+plen, ilistprimes,ilistPrimeCount*INTSIZE);
    printList(primeListNow, totalPrimes);
    */

    fclose(fout);

    return 0;
}
