#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define block_size   32
#define vector_size  1000000000

#define ROWS 300
#define K 400
#define COLS 500
#define INTSIZE sizeof(int)
#define BLOCK_SIZE 16 

__global__ void matMult(int* a, int* b, int* res, int rows, int k, int cols){
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;

    if(r< rows && c< cols){
        for(int x=0; x<k; x++){
            sum += a[r*k +x] + b[x*cols + c]; 
        }
        res[r*cols + c] = sum;
    }
}

int main( void ) { 

    // Set device that we will use for our cuda code
    // It will be either 0 or 1
    hipSetDevice(0);

    srand(time(NULL));

    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    // Express matrix elements as 1 dimension
    int aSize =  ROWS * K * INTSIZE;
    int bSize =  K * COLS* INTSIZE;
    int cSize =  ROWS * COLS * INTSIZE;

    int *a, *b, *c_cpu, *c_gpu;
    hipHostMalloc((void**)&a,aSize);
    hipHostMalloc((void**)&b,bSize);
    hipHostMalloc((void**)&c_cpu,cSize);
    hipHostMalloc((void**)&c_gpu,cSize);

    // Pointers in GPU memory
    int *dev_a;
    int *dev_b;
    int *dev_c;

    // fill the arrays 'a' and 'b' on the CPU
    for(int r=0; r<ROWS; r++){
        for(int c=0; c<K; c++){
            a[ r*K + c] = rand()%10;
        }
    }

    for(int r=0; r<K; r++){
        for(int c=0; c<COLS; c++){
            b[ r*COLS + c ] = rand()%10;
        }
    }

    //
    // CPU Calculation
    //////////////////

    printf("Running sequential job.\n");
    hipEventRecord(start,0);

    // Calculate C in the CPU
    for(int r=0; r<ROWS; r++){
        for(int c=0; c<COLS; c++){

            int sum = 0; 
            for(int k=0; k<K;k++){
                sum +=  a[r*K + k] + b[k*COLS + c];
            }
            c_cpu[r*COLS + c] = sum;
        }
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tSequential Job Time: %.2f ms\n", time);


    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a,  aSize);
    hipMalloc( (void**)&dev_b,  bSize);
    hipMalloc( (void**)&dev_c,  cSize);

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, aSize, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, bSize, hipMemcpyHostToDevice );

    //
    // GPU Calculation
    ////////////////////////
    printf("Running parallel job.\n");

    int gridRows =  (ROWS + BLOCK_SIZE - 1)/ BLOCK_SIZE; 
    int gridCols =  (COLS+ BLOCK_SIZE - 1)/ BLOCK_SIZE; 

    dim3 grids(gridCols, gridRows);
    dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);

    hipEventRecord(start,0);
    matMult<<<grids, blocks>>>(dev_a, dev_b, dev_c, ROWS, K, COLS);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tParallel Job Time: %.2f ms\n", time);

    hipMemcpy( c_gpu, dev_c, cSize, hipMemcpyDeviceToHost);

    // compare the results
    int error = 0;
    for(int r=0; r<ROWS; r++){
        for(int c=0; c<COLS; c++){
            if (c_cpu[r*COLS + c] != c_gpu[r*COLS + c]){
                error = 1;
                break;
            }
        }
    }

    if (error == 0){
        printf ("Correct result. No errors were found.\n");
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c_cpu);
    hipHostFree(c_gpu);

    return 0;
}


