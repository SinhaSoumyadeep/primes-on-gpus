#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define EXECCPU 0

#define block_size   32
#define vector_size  1000000000

#define ROWS 3000
#define K 4000
#define COLS 5000
#define INTSIZE sizeof(unsigned int)
#define BLOCK_SIZE 32 

__global__ void matMult(int* a, int* b, int* res,unsigned  int rows, unsigned int k, unsigned int cols){
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned int sum = 0;

    if(r< rows && c< cols){
        for(int x=0; x<k; x++){
            sum += a[r*k +x] + b[x*cols + c]; 
        }
        res[r*cols + c] = sum;
    }
}

int main( void ) { 

    // Set device that we will use for our cuda code
    // It will be either 0 or 1
    hipSetDevice(1);

    srand(time(NULL));

    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    // Express matrix elements as 1 dimension
    unsigned int aSize =  ROWS * K * INTSIZE;
    unsigned int bSize =  K * COLS* INTSIZE;
    unsigned int cSize =  ROWS * COLS * INTSIZE;

    int *a, *b, *c_cpu, *c_gpu;
    hipHostMalloc((void**)&a,aSize);
    hipHostMalloc((void**)&b,bSize);
    hipHostMalloc((void**)&c_cpu,cSize);
    hipHostMalloc((void**)&c_gpu,cSize);

    // Pointers in GPU memory
    int *dev_a;
    int *dev_b;
    int *dev_c;

    // fill the arrays 'a' and 'b' on the CPU
    for(int r=0; r<ROWS; r++){
        for(int c=0; c<K; c++){
            a[ r*K + c] = rand()%10;
        }
    }

    for(int r=0; r<K; r++){
        for(int c=0; c<COLS; c++){
            b[ r*COLS + c ] = rand()%10;
        }
    }

    //
    // CPU Calculation
    //////////////////

    printf("Running sequential job.\n");
    hipEventRecord(start,0);

    if(EXECCPU){
        // Calculate C in the CPU
        for(unsigned int r=0; r<ROWS; r++){
            for(unsigned int c=0; c<COLS; c++){

                int sum = 0; 
                for(int k=0; k<K;k++){
                    sum +=  a[r*K + k] + b[k*COLS + c];
                }
                c_cpu[r*COLS + c] = sum;
            }
        }
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tSequential Job Time: %.2f ms\n", time);


    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a,  aSize);
    hipMalloc( (void**)&dev_b,  bSize);
    hipMalloc( (void**)&dev_c,  cSize);

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, aSize, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, bSize, hipMemcpyHostToDevice );

    //
    // GPU Calculation
    ////////////////////////
    printf("Running parallel job.\n");

    unsigned int gridRows =  (ROWS + BLOCK_SIZE - 1)/ BLOCK_SIZE; 
    unsigned int gridCols =  (COLS+ BLOCK_SIZE - 1)/ BLOCK_SIZE; 

    dim3 grids(gridCols, gridRows);
    dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);

    hipEventRecord(start,0);
    matMult<<<grids, blocks>>>(dev_a, dev_b, dev_c, ROWS, K, COLS);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tParallel Job Time: %.2f ms\n", time);

    hipMemcpy( c_gpu, dev_c, cSize, hipMemcpyDeviceToHost);

    if(EXECCPU){
        // compare the results
        int error = 0;
        for(unsigned int r=0; r<ROWS; r++){
            for(unsigned int c=0; c<COLS; c++){
                if (c_cpu[r*COLS + c] != c_gpu[r*COLS + c]){
                    error = 1;
                    break;
                }
            }
        }

        if (error == 0){
            printf ("Correct result. No errors were found.\n");

        }
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c_cpu);
    hipHostFree(c_gpu);

    return 0;
}


