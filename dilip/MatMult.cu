
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define block_size   32
#define vector_size  1000000000

#define ROWS 3
#define K 4
#define COLS 5
#define INTSIZE sizeof(int)

int main( void ) { 

    // Set device that we will use for our cuda code
    // It will be either 0 or 1
    hipSetDevice(0);

    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    // Express matrix elements as 1 dimension
    int aSize =  ROWS * K * INTSIZE;
    int bSize =  K * COLS* INTSIZE;
    int cSize =  ROWS * COLS * INTSIZE;

    // Input Arrays and variables
    int *a        = new int [aSize]; 
    int *b        = new int [bSize]; 
    int *c_cpu    = new int [cSize]; 
    //int *c_gpu    = new int [cSize];

    /*
    // Pointers in GPU memory
    int *dev_a;
    int *dev_b;
    int *dev_c;
    */


    // fill the arrays 'a' and 'b' on the CPU
    for(int r=0; r<ROWS; r++){
        for(int c=0; c<K; c++){
            a[ r*K + c] = rand()%10;
        }
    }

    for(int r=0; r<K; r++){
        for(int c=0; c<COLS; c++){
            b[ r*COLS + c ] = rand()%10;
        }
    }

    //
    // CPU Calculation
    //////////////////

    printf("Running sequential job.\n");
    hipEventRecord(start,0);

    // Calculate C in the CPU
    for(int r=0; r<ROWS; r++){
        for(int c=0; c<COLS; c++){

            int sum = 0; 
            for(int k=0; k<K;k++){
                sum +=  a[r*K + k] + b[k*COLS + c];
            }
            c_cpu[r*COLS + c] = sum;
        }
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tSequential Job Time: %.2f ms\n", time);

    free(a);
    free(b);
    free(c_cpu);

    return 0;
}

