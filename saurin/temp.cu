
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void calcPrimes(int *d_IL, int *d_PL, int numOfPrimes, int lenInputList);

#define LEN_IL 100
#define LEN_INITIAL_PRIMES 4
#define THREADS_PER_BLOCK 32

int main() {
	int *IL, *PL;
	int *d_IL, *d_PL;
	int count = 0;

	int size_IL = LEN_IL * sizeof(int);
	int size_PL = LEN_INITIAL_PRIMES * sizeof(int);

	// Space for device copies:
	hipMalloc((void **) &d_IL, size_IL);
	hipMalloc((void **) &d_PL, size_PL);

	// Space for host copies:
	IL = (int*) malloc(LEN_IL * sizeof(int));
	PL = (int*) malloc(LEN_INITIAL_PRIMES * sizeof(int));

	//Initialize Primes list:
	PL[0] = 2;
	PL[1] = 3;
	PL[2] = 5;
	PL[3] = 7;

	// Initialize Input list: 0 -> Not prime:
	for(int i=0; i<LEN_IL; i++) {
		IL[i] = 1;
	}

	// Copying the data to the device (GPU):
	hipMemcpy(d_IL, IL, size_IL, hipMemcpyHostToDevice);
	hipMemcpy(d_PL, PL, size_PL, hipMemcpyHostToDevice);

	// Launching the kernel:
	calcPrimes<<<(LEN_INITIAL_PRIMES/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>> (d_IL, d_PL, LEN_INITIAL_PRIMES, LEN_IL);

	int *result = (int*) malloc(size_IL);

	// Copy the result back to the host:
	hipMemcpy(result, d_IL, size_IL, hipMemcpyDeviceToHost);

	// Output the existing primes:
	printf("\nExisting (old) Primes List:\n");
	for(int i=0; i<LEN_INITIAL_PRIMES; i++) {
		printf("%d\t", PL[i]);
	}
	printf("\n");

	// Output the new calculated primes: (1 -> Prime)
	printf("\nNew Primes List:\n");
	for(int i=PL[LEN_INITIAL_PRIMES-1]+1; i < LEN_IL; i++) {
		if(result[i] == 1) {
			printf("%d\t", i);
			count++;
		}
	}
	printf("\n");
	printf("Number of primes found = %d\n\n", count);

	// Free memory:
	free(IL);
	free(PL);
	free(result);
	hipFree(d_IL);
	hipFree(d_PL);

	return 0;
}

	
__global__ void calcPrimes(int *d_IL, int *d_PL, int numOfPrimes, int lenInputList) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < numOfPrimes) {
		for(int i = d_PL[numOfPrimes-1]+1; i < lenInputList; i++) {
			if(i % d_PL[index] == 0) {
				d_IL[i] = 0;
			}
		}
	}
}


