
#include <hip/hip_runtime.h>
#include<stdio.h>

int generateInitialPrimes(int *intialTempArray, int initialPrimesRange);
__global__ void calcPrimes(int *d_IL, int *d_PL, int numOfPrimes, int lenInputList);

#define LEN_IL 10000
#define LEN_INITIAL_PRIMES 100
#define THREADS_PER_BLOCK 32

int main() {
	int *IL, *PL;
	int *d_IL, *d_PL;
	int count = 0;

	// Space for host copies:
	IL = (int*) malloc(LEN_IL * sizeof(int));
	PL = (int*) malloc(LEN_INITIAL_PRIMES * sizeof(int));

	int numOfInitialPrimes = generateInitialPrimes(PL, LEN_INITIAL_PRIMES);

	int size_IL = LEN_IL * sizeof(int);
	int size_PL = numOfInitialPrimes * sizeof(int);

	//Initialize Primes list:
	/*
	PL[0] = 2;
	PL[1] = 3;
	PL[2] = 5;
	PL[3] = 7;
	*/
	// Initialize Input list: 0 -> Not prime:
	for(int i=0; i<LEN_IL; i++) {
		IL[i] = 1;
	}

	// Space for device copies:
	hipMalloc((void **) &d_IL, size_IL);
	hipMalloc((void **) &d_PL, size_PL);

	// Copying the data to the device (GPU):
	hipMemcpy(d_IL, IL, size_IL, hipMemcpyHostToDevice);
	hipMemcpy(d_PL, PL, size_PL, hipMemcpyHostToDevice);

	// Launching the kernel:
	calcPrimes<<<(numOfInitialPrimes/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>> (d_IL, d_PL, numOfInitialPrimes, LEN_IL);

	int *result = (int*) malloc(size_IL);

	// Copy the result back to the host:
	hipMemcpy(result, d_IL, size_IL, hipMemcpyDeviceToHost);

	// Output the existing primes:
	printf("\nExisting (old) Primes List:\n");
	for(int i=0; i<numOfInitialPrimes; i++) {
		printf("%d\t", PL[i]);
	}
	printf("\n");

	// Output the new calculated primes: (1 -> Prime)
	printf("New Primes List:\n");
	for(int i=PL[numOfInitialPrimes-1]+1; i < LEN_IL; i++) {
		if(result[i] == 1) {
			printf("%d\t", i);
			count++;
		}
	}
	printf("\n");
	printf("Number of new primes found = %d\n\n", count);

	// Free memory:
	free(IL);
	free(PL);
	free(result);
	hipFree(d_IL);
	hipFree(d_PL);

	return 0;
}



// Returns: Count of primes
int generateInitialPrimes(int *intialTempArray, int initialPrimesRange) {
	int primesCount = 0;
	//int intialTempArray[initialPrimesRange];
	//intialTempArray = (int*) malloc(LEN_INITIAL_PRIMES * sizeof(int));
	
	// Initialize array with all 1's:
	for(int i=0; i<initialPrimesRange; i++) {
		intialTempArray[i] = 1;
	}

	// Make non-primes as '0':
	for(int i=2; i*i <= initialPrimesRange; i++) {
		for(int j=2*i; j <= initialPrimesRange; j=j+i) {
				intialTempArray[j] = 0;
		}
	}
	
	// Print the initial primes:
	printf("\n Initial Primes are: \n");
	for(int i=2; i<=initialPrimesRange; i++) {
		int num = intialTempArray[i];
		if(num == 1) {
			printf("%d  ", i);	
			primesCount++;
		}
	}
	printf("\n\nCount of initial primes = %d\n", primesCount);
	return primesCount;
}


	
__global__ void calcPrimes(int *d_IL, int *d_PL, int numOfPrimes, int lenInputList) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < numOfPrimes) {
		for(int i = d_PL[numOfPrimes-1]+1; i < lenInputList; i++) {
			if(i % d_PL[index] == 0) {
				d_IL[i] = 0;
			}
		}
	}
}


