
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

int generateInitialPrimes(int *intialTempArray, int **PL, int initialPrimesRange);
__global__ void calcPrimes(int *d_IL, int *d_PL, int numOfPrimes, int lenInputList);

#define LEN_IL 1000000
#define LEN_INITIAL_PRIMES 1000
#define THREADS_PER_BLOCK 32

int main() {
	int *IL = NULL, *PL = NULL, *tempPL = NULL;
	int *d_IL = NULL, *d_PL = NULL;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	clock_t t;
	//int count = 0;

	t = clock();
	int numOfInitialPrimes = generateInitialPrimes(tempPL, &PL, LEN_INITIAL_PRIMES);
	t = clock() - t;
	double time_taken = ((double)t)/CLOCKS_PER_SEC; 			// in seconds

	// Print the initial range of primes calculated in the CPU, which will be passed to the GPU:
	printf("\nThe initial primes calculated are:\n");
	for(int i=0; i < numOfInitialPrimes; i++) {
		printf("%d  ", PL[i]);
	}
	printf("\nNumber of initial primes = %d\n\n", numOfInitialPrimes);

	// Space for host copies:
	IL = (int*) malloc(LEN_IL * sizeof(int));
	//PL = (int*) malloc(LEN_INITIAL_PRIMES * sizeof(int));		   		// Allocated in the generate function instead


	int size_IL = LEN_IL * sizeof(int);
	int size_PL = numOfInitialPrimes * sizeof(int);

	// Initialize Input list: 0 -> Not prime:
	for(int i=0; i<LEN_IL; i++) {
		IL[i] = 1;
	}

	// Space for device copies:
	hipMalloc((void **) &d_IL, size_IL);
	hipMalloc((void **) &d_PL, size_PL);

	// Copying the data to the device (GPU):
	hipMemcpy(d_IL, IL, size_IL, hipMemcpyHostToDevice);
	hipMemcpy(d_PL, PL, size_PL, hipMemcpyHostToDevice);			

	// Launching the kernel and measuring the time taken:
	hipEventRecord(start, 0);
	calcPrimes<<<(numOfInitialPrimes/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>> (d_IL, d_PL, numOfInitialPrimes, LEN_IL);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// Space allocated to store the modified form of input array, with marking for prime and non-prime:
	int *result = (int*) malloc(size_IL);

	// Copy the result back to the host:
	hipMemcpy(result, d_IL, size_IL, hipMemcpyDeviceToHost);

	// Extract indexes of primes in 'result' to get the actual new prime numbers:
	printf("********* New Primes List **********\n");
	int *newPrimes = (int*)malloc(LEN_IL / 4 * sizeof(int));			// Arbitrary size; which is '1/4th' of numbers list size
	int newPrimesCount = 0;
	for(int i=LEN_INITIAL_PRIMES; i<LEN_IL; i++) {
		int num = result[i];
		if(num == 1) {
			newPrimes[newPrimesCount] = num;
			newPrimesCount++;
			printf("%d  ", i);
		}
	}
	printf("\n\nNumber of new primes found = %d\n\n", newPrimesCount);

	printf("Time taken to find initial primes on CPU = %f ms\n", time_taken * 1000);
	printf("Parallel Job time for current iteration = %f ms\n\n", time);

	// Free memory:
	hipFree(d_IL);
	hipFree(d_PL);
	free(IL);
	free(PL);
	free(result);
	free(newPrimes);

	return 0;
}


// Generate initial prime numbers in the CPU:
// Returns: Number of primes found from 1 to 'LEN_INITIAL_PRIMES' 
int generateInitialPrimes(int *intialTempArray, int **PL, int initialPrimesRange) {
	int primesCount = 0;
	//int intialTempArray[initialPrimesRange];
	intialTempArray = (int*) malloc(LEN_INITIAL_PRIMES * sizeof(int));
	*PL = (int*) malloc(LEN_INITIAL_PRIMES / 2 * sizeof(int));				// Taking half size of initial (full) primes array
	
	// Initialize array with all 1's:
	for(int i=0; i < initialPrimesRange; i++) {
		intialTempArray[i] = 1;
	}

	// Make non-primes as '0':
	for(int i=2; i*i <= initialPrimesRange; i++) {
		for(int j=2*i; j <= initialPrimesRange; j=j+i) {
				intialTempArray[j] = 0;
		}
	}
	
	// Store the actual primes in a new array which will be copied later to the device (converting 'prime num indexes' to 'prime numbers') :
	for(int i=2; i<=initialPrimesRange; i++) {
		if(intialTempArray[i] == 1) {
			(*PL)[primesCount] = i;
			primesCount++;
		}
	}

	free(intialTempArray);
	return primesCount;
}



// GPU Kernel (Parallel Processing):
__global__ void calcPrimes(int *d_IL, int *d_PL, int numOfPrimes, int lenInputList) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < numOfPrimes) {
		for(int i = d_PL[numOfPrimes-1]+1; i < lenInputList; i++) {
			if(i % d_PL[index] == 0) {
				d_IL[i] = 0;
			}
		}
	}
}


