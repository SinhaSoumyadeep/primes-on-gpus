
#include <hip/hip_runtime.h>
#include <stdio.h>

int generateInitialPrimes(int *intialTempArray, int **PL, int initialPrimesRange);
__global__ void calcPrimes(int *d_IL, int *d_PL, int numOfPrimes, int lenInputList);

#define LEN_IL 10000
#define LEN_INITIAL_PRIMES 100
#define THREADS_PER_BLOCK 32

int main() {
	int *IL = NULL, *PL = NULL, *tempPL = NULL;
	int *d_IL, *d_PL;
	//int count = 0;

	// Space for host copies:
	IL = (int*) malloc(LEN_IL * sizeof(int));
	PL = (int*) malloc(LEN_INITIAL_PRIMES * sizeof(int));

	int numOfInitialPrimes = generateInitialPrimes(tempPL, &PL, LEN_INITIAL_PRIMES);

	// Print the initial range of primes calculated in the CPU, which will be passed to the GPU:
	printf("\nThe initial primes calculated are:\n");
	for(int i=0; i < numOfInitialPrimes; i++) {
		printf("%d  ", PL[i]);
	}
	printf("\n\nCount of initial primes = %d\n\n", numOfInitialPrimes);


	int size_IL = LEN_IL * sizeof(int);
	int size_PL = numOfInitialPrimes * sizeof(int);

	// Initialize Input list: 0 -> Not prime:
	for(int i=0; i<LEN_IL; i++) {
		IL[i] = 1;
	}

	// Space for device copies:
	hipMalloc((void **) &d_IL, size_IL);
	hipMalloc((void **) &d_PL, size_PL);

	// Copying the data to the device (GPU):
	hipMemcpy(d_IL, IL, size_IL, hipMemcpyHostToDevice);
	hipMemcpy(d_PL, PL, size_PL, hipMemcpyHostToDevice);			/// NEEDS CORRECTION, 'PL' has length of 'LEN_INITIAL_PRIMES' and not 'size_PL'

	// Launching the kernel:
	calcPrimes<<<(numOfInitialPrimes/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>> (d_IL, d_PL, numOfInitialPrimes, LEN_IL);  // CHECK if it should be 'numOfInitialPrimes' or 'LEN_INITIAL_PRIMES'

	// Space allocated to store the modified form of input array, with marking for prime and non-prime:
	int *result = (int*) malloc(size_IL);

	// Copy the result back to the host:
	hipMemcpy(result, d_IL, size_IL, hipMemcpyDeviceToHost);

	// Extract indexes of primes in 'result' to get the actual new prime numbers:
	printf("New Primes List:\n");
	int *newPrimes = (int*)malloc(LEN_IL / 4 * sizeof(int));
	int newPrimesCount = 0;
	for(int i=LEN_INITIAL_PRIMES; i<LEN_IL; i++) {
		int num = result[i];
		if(num == 1) {
			newPrimes[newPrimesCount] = num;
			newPrimesCount++;
			printf("%d  ", i);
		}
	}
	printf("\nNumber of new primes found = %d\n\n", newPrimesCount);



	/* Output the existing primes:										// SECTION NEEDS CHANGES
	printf("\nExisting (old) Primes List:\n");
	for(int i=0; i<numOfInitialPrimes; i++) {
		printf("%d\t", PL[i]);
	}
	printf("\n");
	*/

/*
	// Output the new calculated primes: (1 -> Prime)					// SECTION NEEDS CHANGES
	printf("New Primes List:\n");
	for(int i=PL[numOfInitialPrimes-1]+1; i < LEN_IL; i++) {
		if(result[i] == 1) {
			printf("%d\t", i);
			count++;
		}
	}
	printf("\n");
	printf("Number of new primes found = %d\n\n", count);
*/
	// Free memory:
	free(IL);
	free(PL);
	free(result);
	hipFree(d_IL);
	hipFree(d_PL);

	return 0;
}



// Returns: Count of primes
int generateInitialPrimes(int *intialTempArray, int **PL, int initialPrimesRange) {
	int primesCount = 0;
	//int intialTempArray[initialPrimesRange];
	intialTempArray = (int*) malloc(LEN_INITIAL_PRIMES * sizeof(int));
	*PL = (int*) malloc(LEN_INITIAL_PRIMES / 2 * sizeof(int));				// Taking half size of initial (full) primes array
	
	// Initialize array with all 1's:
	for(int i=0; i < initialPrimesRange; i++) {
		intialTempArray[i] = 1;
	}

	// Make non-primes as '0':
	for(int i=2; i*i <= initialPrimesRange; i++) {
		for(int j=2*i; j <= initialPrimesRange; j=j+i) {
				intialTempArray[j] = 0;
		}
	}
	
	// Store the actual primes in a new array which will be copied later to the device (converting 'prime num indexes' to 'prime numbers') :
	for(int i=2; i<=initialPrimesRange; i++) {
		if(intialTempArray[i] == 1) {
			(*PL)[primesCount] = i;
			primesCount++;
		}
	}
	return primesCount;
}


	
__global__ void calcPrimes(int *d_IL, int *d_PL, int numOfPrimes, int lenInputList) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < numOfPrimes) {
		for(int i = d_PL[numOfPrimes-1]+1; i < lenInputList; i++) {
			if(i % d_PL[index] == 0) {
				d_IL[i] = 0;
			}
		}
	}
}


