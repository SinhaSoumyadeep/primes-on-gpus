#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c)
{
	int i= blockIdx.x*blockDim.x+threadIdx.x;
	
		c[i]= a[i]+b[i];

}

int main()
{
	// host pointers
	int *a;
	int *b;
	int *c;
	//device pointers
	int *d_a;
	int *d_b;
	int *d_c;

	a=(int *)malloc(10*sizeof(int));
	b=(int *)malloc(10* sizeof(int));
	c=(int *)malloc(10*sizeof(int));
	int i=0;
	for(i=0;i<10;i++)
	{
		a[i]=i;
		b[i]=i+1;
	}
	

	hipMalloc(&d_a, 10*sizeof(int));
	hipMalloc(&d_b, 10*sizeof(int));
	hipMalloc(&d_c, 10*sizeof(int));

	hipMemcpy(d_a,a,10*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,10*sizeof(int),hipMemcpyHostToDevice);


	add<<<1,10>>>(d_a,d_b,d_c);

	hipMemcpy(c,d_c,10*sizeof(int),hipMemcpyDeviceToHost);


	int j=0;

	for(j=0;j<10;j++)
	{
		printf("%d\n",c[j] );
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return 0;
}
