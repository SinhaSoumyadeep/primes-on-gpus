#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

__global__ void prime_generator(int *input,int *prime_list,int *total_input,int *seed)
{

	printf("-------XXXXXX>>> %d\n",seed[0]);
	int i= blockIdx.x * blockDim.x + threadIdx.x;
	int primeno= prime_list[i];
	int total=seed[0]*seed[0];
	for(int k=seed[0];k<total;k++)
	{
		if(k%primeno==0)
		{
			input[k]=1;
			
		
		}
	

	}
	


}

int main()
{
	int total_input=100000000;
	int *input;
	int n= 10 ;// seed prime list.
	int calculate_upto=pow(n,2);
	int *primelist;
	input=(int *)malloc(total_input*sizeof(int));
	primelist=(int *)malloc(total_input*sizeof(int));
        memset(input,-1,total_input*sizeof(int));
	for(int j=0;j<calculate_upto;j++)
	{
		input[j]=0;
	}

	
	for (int p=2; p*p<=n; p++)
    {
        
        if (input[p] == 0)
        {
            for (int i=p*2; i<=n; i += p)
                input[i] = 1;
        }
    }




int i=0;
    for (int p=2; p<=n; p++){

    		 
       if (input[p]==0)
       {
       		
       		primelist[i]=p;
       		i++;
       }

     

   }


int *d_input;
int *d_prime_list;
int *h_pl;
int *d_total_input;
int *d_seed;

h_pl=(int *)malloc(total_input*sizeof(int));

hipMalloc(&d_input,total_input*sizeof(int));
hipMalloc(&d_prime_list,total_input*sizeof(int));
hipMalloc(&d_total_input,sizeof(int));
hipMalloc(&d_seed,sizeof(int));

hipMemcpy(d_total_input,&total_input,sizeof(int),hipMemcpyHostToDevice);
//cudaMemcpy(d_input,input,total_input*sizeof(int),cudaMemcpyHostToDevice);


while(n<=total_input){

printf("inside loop\n");
if(hipMemcpy(d_input,input,total_input*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
{
		printf("not able to copy memory\n");
}
if(hipMemcpy(d_prime_list,primelist,total_input*sizeof(int),hipMemcpyHostToDevice) != hipSuccess)
{
	printf("not able to copy memory 2\n");
}
if(hipMemcpy(d_seed,&n,sizeof(int),hipMemcpyHostToDevice) != hipSuccess)
{

	printf(" not able to copy memory\n");
}

prime_generator<<<5,500>>>(d_input,d_prime_list,d_total_input,d_seed);

hipError_t err = hipGetLastError();
if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));

/*
if(cudaMemcpy(h_pl,d_prime_list,total_input*sizeof(int),cudaMemcpyDeviceToHost)!=cudaSuccess)
{
	printf("not able to copy memory!!\n");

}
*/

if(hipMemcpy(input,d_input,total_input*sizeof(int),hipMemcpyDeviceToHost)!=hipSuccess)
{
	printf(" hello not able to copy memory::\n");

}
printf("------------>> %d\n",i);

for(int p=n;p<total_input;p++)
{

//	printf("%d ----> %d\n",p,input[p]);
 	if(input[p]==0){
	primelist[i]=p;
	i++;
	}
}		
for(int p=0;p<i;p++)
{
       printf("%d\n",primelist[p]);

}
n=n*n;
printf("################  %d\n",n);
if(pow(n,2)>=total_input){
for(int m=n;m<total_input;m++) input[m]=0;
}
else
{
for(int m=n;m<pow(n,2);m++) input[m]=0;
}

}
/*for(int p=0;p<i;p++)
{
	printf("%d\n",primelist[p]);

}*/



	return 0;
 
 }
