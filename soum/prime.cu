#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

__global__ void prime_generator(int *input,int *prime_list,int *total_input)
{

	int i= blockIdx.x * blockDim.x + threadIdx.x;
	int primeno= prime_list[i];

	for(int k=10;k<total_input[0];k++)
	{
		if(k%primeno==0)
		{
			input[k]=1;
				
		}

	}
	


}

int main()
{
	int total_input=100;
	int *input;
	int n= 10 ;// seed prime list.
	int *primelist;
	input=(int *)malloc(total_input*sizeof(int));
	primelist=(int *)malloc(total_input*sizeof(int));

	for(int j=0;j<total_input;j++)
	{
		input[j]=0;
	}

	
	for (int p=2; p*p<=n; p++)
    {
        
        if (input[p] == 0)
        {
            for (int i=p*2; i<=n; i += p)
                input[i] = 1;
        }
    }




int i=0;
    for (int p=2; p<=n; p++){

    		 
       if (input[p]==0)
       {
       		
       		primelist[i]=p;
       		i++;
       }

     

   }


int *d_input;
int *d_prime_list;
int *h_pl;
int *d_total_input;

h_pl=(int *)malloc(i*sizeof(int));

hipMalloc(&d_input,total_input*sizeof(int));



hipMalloc(&d_prime_list,total_input*sizeof(int));
hipMalloc(&d_total_input,sizeof(int));


hipMemcpy(d_input,input,total_input*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_prime_list,primelist,i*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_total_input,&total_input,sizeof(int),hipMemcpyHostToDevice);

prime_generator<<<1,4>>>(d_input,d_prime_list,d_total_input);

hipMemcpy(h_pl,d_prime_list,i*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(input,d_input,total_input*sizeof(int),hipMemcpyDeviceToHost);

for(int p=2;p<total_input;p++)
{
	if(input[p]==1)
	continue;
        		
	printf(" %d\n",p);
}







	return 0;
 
 }


