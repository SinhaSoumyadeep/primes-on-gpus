#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

__global__ void prime_generator(int *input,int *prime_list,int *total_input,int *seed)
{

	int i= blockIdx.x * blockDim.x + threadIdx.x;
	int primeno= prime_list[i];
	
	for(int k=seed[0];k<total_input[0];k++)
	{
		if(k%primeno==0)
		{
			input[k]=1;
		
		}
	

	}
	


}

int main()
{
	int total_input=1e6;
	int *input;
	int n= 1000 ;// seed prime list.
	int *primelist;
	input=(int *)malloc(total_input*sizeof(int));
	primelist=(int *)malloc(total_input*sizeof(int));

	for(int j=0;j<total_input;j++)
	{
		input[j]=0;
	}

	
	for (int p=2; p*p<=n; p++)
    {
        
        if (input[p] == 0)
        {
            for (int i=p*2; i<=n; i += p)
                input[i] = 1;
        }
    }




int i=0;
    for (int p=2; p<=n; p++){

    		 
       if (input[p]==0)
       {
       		
       		primelist[i]=p;
       		i++;
       }

     

   }


int *d_input;
int *d_prime_list;
int *h_pl;
int *d_total_input;
int *d_seed;

h_pl=(int *)malloc(total_input*sizeof(int));

hipMalloc(&d_input,total_input*sizeof(int));
hipMalloc(&d_prime_list,total_input*sizeof(int));
hipMalloc(&d_total_input,sizeof(int));
hipMalloc(&d_seed,sizeof(int));

hipMemcpy(d_total_input,&total_input,sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_input,input,total_input*sizeof(int),hipMemcpyHostToDevice);



hipMemcpy(d_prime_list,primelist,total_input*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_seed,&n,sizeof(int),hipMemcpyHostToDevice);

prime_generator<<<1,1000>>>(d_input,d_prime_list,d_total_input,d_seed);

hipMemcpy(h_pl,d_prime_list,total_input*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(input,d_input,total_input*sizeof(int),hipMemcpyDeviceToHost);


for(int p=2;p<total_input;p++)
{
 	if(input[p]==1)
	continue;
        		
	primelist[i]=p;       
	i++;
}		
	
for(int p=0;p<total_input;p++)
{
	printf("%d\n",primelist[p]);

}







	return 0;
 
 }


