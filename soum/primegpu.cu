#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include<hip/hip_runtime.h>
#define INPUT_SIZE 100000000
#define PRIME_RANGE 1000000
#define BLOCK_SIZE 32
typedef unsigned long long int uint64_c;

void initializeInput(char* , int );
int generate_seed_primes(char*, int*, uint64_c);
void copy_seed_primes(uint64_c *,int *,int);
void print_primelist(uint64_c *, uint64_c);
void print_inputlist(uint64_c *input_list,uint64_c range);
void initializing_inputlist(uint64_c *input_list, uint64_c start,uint64_c range);
void calculatePrime(uint64_c* , uint64_c* , uint64_c ,uint64_c);
void appending_prime(uint64_c* input_list, uint64_c* prime_list, uint64_c range, uint64_c prev_number_of_primes, uint64_c number_of_primes);
uint64_c counting_primes(uint64_c*, uint64_c, uint64_c);

//KERNAL CODE GOES HERE!!


__global__ void prime_generator(uint64_c* d_input_list, uint64_c* d_prime_list, uint64_c* d_range,uint64_c* d_number_of_primes)
{


    int p= blockIdx.x * blockDim.x + threadIdx.x;
    int prime = d_prime_list[p];

        for(uint64_c i=0;i<d_range[0];i++){


                        if(d_input_list[i] % prime ==0)
                        {
                                d_input_list[i]=0;
                                
                               
                        }
                
        }



}


//KERNAL CODE ENDS HERE!!!


int main()
{


 hipSetDevice(1);

// This code is just to generate the seed prime numbers
        int input_size=100;
        char *input;
        uint64_c n= 10 ;// seed prime list.
        int *seed_primelist;
        input=(char *)malloc(input_size*sizeof(char));
        initializeInput(input, input_size);
        seed_primelist=(int *)malloc(input_size*sizeof(int));
        int num_of_seed = generate_seed_primes(input,seed_primelist,n);
    
        uint64_c* input_list;
        uint64_c* prime_list;
        uint64_c number_of_primes= num_of_seed; 
        prime_list=(uint64_c *)malloc(number_of_primes*sizeof(uint64_c));
        copy_seed_primes(prime_list,seed_primelist,num_of_seed);

        uint64_c* d_input_list;
        uint64_c* d_prime_list;
        uint64_c* d_number_of_primes;
	uint64_c* d_range;

       
            for(int i=0;i<3;i++){


                uint64_c start=n;
                uint64_c end=pow(n,2);
                printf("CALCULATING PRIMES FROM 0 - %llu\n",end);
                uint64_c range=end-start;
                input_list=(uint64_c *)malloc(range*sizeof(uint64_c));
                initializing_inputlist(input_list,start,range);
		

		if(hipMalloc((void **)&d_input_list,range*sizeof(uint64_c))!=hipSuccess)
                {
                    printf("Error:  1\n");
                }
                
                if(hipMemcpy(d_input_list,input_list,range*sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                {
                    printf("copy Error:  1\n");
                }

                if(hipMalloc((void **)&d_prime_list,number_of_primes*sizeof(uint64_c))!=hipSuccess)
                {
                    printf("Error:  2\n");
                }

                if(hipMemcpy(d_prime_list,prime_list,number_of_primes*sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                {
                    printf("copy Error:  2\n");
                }

                if(hipMalloc((void **)&d_range,sizeof(uint64_c))!=hipSuccess)
                {
                    printf("Error:  3\n");
                }
                
                if(hipMemcpy(d_range,&range,sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                {
                    printf("copy Error:  3\n");
                }

                if(hipMalloc((void **)&d_number_of_primes,sizeof(uint64_c))!=hipSuccess)
                {
                    printf("Error:  4\n");
                }
                
                if(hipMemcpy(d_number_of_primes,&number_of_primes,sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                {
                    printf("copy Error:  4\n");
                }







              
             //   calculatePrime(input_list,prime_list,range,number_of_primes);

		prime_generator<<<43,32>>>(d_input_list,d_prime_list,d_range,d_number_of_primes);

		if(hipMemcpy(input_list,d_input_list,range*sizeof(uint64_c),hipMemcpyDeviceToHost)!=hipSuccess)
                {
                    printf("copy Host Error:  1\n");
			exit(0);
                }





                
                //print_inputlist(input_list,range);
                uint64_c previous_number_of_primes= number_of_primes;
                number_of_primes = counting_primes(input_list, range, number_of_primes)+previous_number_of_primes;
                printf("THE NUMBER OF PRIMES ARE: %llu\n",number_of_primes);
                prime_list=(uint64_c *)realloc(prime_list,number_of_primes*sizeof(uint64_c));
                appending_prime(input_list, prime_list,  range, previous_number_of_primes, number_of_primes);
                print_primelist(prime_list,number_of_primes);
                n=pow(n,2);
                printf("******************************************\n\n");

            }







    
        return 0;

 }



int generate_seed_primes(char *input,int *primelist, uint64_c n)
{
    for (int p=2; p*p<=n; p++)
    {
         if (input[p] == 'P')
        {
            for (int i=p*2; i<=n; i += p)
                input[i] = 'N';
        }
    }




int i=0;
    for (int p=2; p<=n; p++){


       if (input[p]=='P')
       {

          primelist[i]=p;
          i++;
       }



   }

   return i;

}

void initializeInput(char *input, int input_size)
{

    for(int i=0;i<input_size;i++)
    {
        input[i]='P';
    }

}

void initializing_inputlist(uint64_c *input_list, uint64_c start,uint64_c range){

        for(uint64_c i=0;i<range;i++)
        {
                input_list[i]=start+i;

        }


}

void print_inputlist(uint64_c *input_list,uint64_c range)
{

        for(uint64_c i=0;i<range;i++)
        {
                printf("%llu\t--->\t%llu\n", i,input_list[i]);
        }


}

void print_primelist(uint64_c *prime_list,uint64_c number_of_primes)
{

        for(uint64_c i=0;i<number_of_primes;i++)
        {
                printf("%llu\n",prime_list[i]);
        }


}


void copy_seed_primes(uint64_c *prime_list,int * seed_primelist,int num_of_seed)
{
        

        for(int i=0;i<num_of_seed;i++)
        {
                prime_list[i]=seed_primelist[i];
                
        }

}



void calculatePrime(uint64_c* input_list, uint64_c* prime_list, uint64_c range,uint64_c number_of_primes)
{
        
       // print_primelist(prime_list,number_of_primes);
            for(uint64_c i=0;i<range;i++)
        {
                for(uint64_c j=0;j<number_of_primes;j++){


                

                        if(input_list[i] % prime_list[j]==0)
                        {
                                input_list[i]=0;
                                
                               
                        }



                }
        }
        

}

void appending_prime(uint64_c* input_list, uint64_c* prime_list, uint64_c range, uint64_c prev_number_of_primes, uint64_c number_of_primes)
 {

                for(uint64_c i=0;i<range;i++)
                {
                        if(input_list[i]>0){
                            //printf("XXXXXXXXX>>> %llu\n", input_list[i]);
                            prime_list[prev_number_of_primes]=input_list[i];
                            prev_number_of_primes++;
                        }

                }
               // printf("the number he he ha ha... %llu\n",prev_number_of_primes);

                //exit(0);
                

 }

uint64_c counting_primes(uint64_c* input_list,uint64_c range,uint64_c number_of_primes)
{  
    
    int prime=0;

    for(uint64_c i=0;i<range;i++)
    {
        if(input_list[i]>0)
        {
            prime++;
        }
    }

    return prime;

}

