#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include<hip/hip_runtime.h>

#define INPUT_SIZE 100000000
#define PRIME_RANGE 10000000
typedef unsigned long long int uint64_c;


int generate_seed_primes(int*, int*, uint64_c);
void copy_seed_primes(uint64_c *,int *,int);
void print_primelist(uint64_c *, uint64_c);
void print_inputlist(uint64_c *);
void initializing_inputlist(uint64_c *);
void memsetting_range_of_input(uint64_c *,uint64_c);
void calculatePrime(uint64_c*, uint64_c*, uint64_c, uint64_c, uint64_c);
uint64_c appending_prime(uint64_c*, uint64_c*, uint64_c, uint64_c, uint64_c);


//KERNAL CODE GOES HERE!!





//KERNAL CODE ENDS HERE!!!


int main()
{

// This code is just to generate the seed prime numbers
        int input_size=100;
        int *input;
        uint64_c n= 10 ;// seed prime list.
        int *seed_primelist;
        input=(int *)malloc(input_size*sizeof(int));
        seed_primelist=(int *)malloc(input_size*sizeof(int));
        int num_of_seed = generate_seed_primes(input,seed_primelist,n);


// seed prime list code ends here.


//Starting code for gpu.
        //declaring host variables.

        // declaring the ranges of the input size and the primes to be generated.

        uint64_c total_input_size = INPUT_SIZE;
        printf("TOTAL INPUT SIZE IS: %llu\n",total_input_size);
        uint64_c prime_range = PRIME_RANGE;
        printf("THE PRIMES WILL BE GENERATED FROM 0 - %llu\n",prime_range);

        printf("-------------------------------------------------------------------------\n\n\n");
        // creating the host array of input-list and primelist.
        uint64_c *input_list;
        uint64_c *prime_list;
        uint64_c number_of_primes= num_of_seed; //initializing the number of primes to the number of seed primes.
        input_list=(uint64_c *)malloc(total_input_size * sizeof(uint64_c));

        //setting all the values of the input list to -1.
        initializing_inputlist(input_list);

        prime_list=(uint64_c *)malloc(prime_range * sizeof(uint64_c));

        //copying the seed primes in prime_list.
                copy_seed_primes(prime_list,seed_primelist,num_of_seed);

         

        //creating the device array of input list and primelist
        uint64_c *device_input_list;
        uint64_c *device_prime_list;
        uint64_c *device_previous_range;
        uint64_c *device_max_prime_range;
        uint64_c *device_number_of_primes;


        //allocating memory in gpu.

        if(hipMalloc((void** )&device_input_list,total_input_size * sizeof(uint64_c))!=hipSuccess)
        {
                printf("ERROR: CANNOT ALLOCATE MEMORY IN GPU FOR INPUT LIST\n");
                exit(0);
        }

        if(hipMalloc((void** )&device_prime_list,prime_range * sizeof(uint64_c))!=hipSuccess)
        {
                printf("ERROR: CANNOT ALLOCATE MEMORY IN GPU FOR PRIME LIST\n");
                hipFree(device_input_list);
                exit(0);
        }

        if(hipMalloc((void** )&device_previous_range,sizeof(uint64_c))!=hipSuccess)
        {
                printf("ERROR: CANNOT ALLOCATE MEMORY IN GPU FOR PREVIOUS RANGE\n");
                hipFree(device_input_list);
                hipFree(device_prime_list);
                exit(0);
        }

        if(hipMalloc((void** )&device_max_prime_range,sizeof(uint64_c))!=hipSuccess)
        {
                printf("ERROR: CANNOT ALLOCATE MEMORY IN GPU FOR MAX PRIME RANGE\n");
                hipFree(device_input_list);
                hipFree(device_prime_list);
                hipFree(device_previous_range);
                exit(0);
        }

        if(hipMalloc((void** )&device_number_of_primes,sizeof(uint64_c))!=hipSuccess)
        {
                printf("ERROR: CANNOT ALLOCATE MEMORY IN GPU FOR NUMBER OF PRIMES\n");
                hipFree(device_input_list);
                hipFree(device_prime_list);
                hipFree(device_previous_range);
                hipFree(device_max_prime_range);
                exit(0);
        }

        //allocating memory in gpu completed.


        //copying input list and prime list from host to device.

        if(hipMemcpy(device_input_list,input_list,total_input_size * sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
        {
                printf("ERROR: CANNOT COPY INPUT LIST FROM HOST TO DEVICE\n");
                hipFree(device_input_list);
                hipFree(device_prime_list);
                hipFree(device_previous_range);
                hipFree(device_max_prime_range);
                hipFree(device_number_of_primes);
                exit(0);
        } 

        if(hipMemcpy(device_prime_list,prime_list,prime_range * sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
        {
                printf("ERROR: CANNOT COPY PRIME LIST FROM HOST TO DEVICE\n");
                hipFree(device_input_list);
                hipFree(device_prime_list);
                hipFree(device_previous_range);
                hipFree(device_max_prime_range);
                hipFree(device_number_of_primes);
                exit(0);
        } 

        //copying input list and prime list from host to device completed.

                while(n<PRIME_RANGE){

                        //copying number of primes generated.
                        if(hipMemcpy(device_number_of_primes,&number_of_primes,sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                        {
                                printf("ERROR: CANNOT COPY NUMBER OF PRIMES FROM HOST TO DEVICE\n");
                                hipFree(device_input_list);
                                hipFree(device_prime_list);
                                hipFree(device_previous_range);
                                hipFree(device_max_prime_range);
                                hipFree(device_number_of_primes);
                                exit(0);
                        } 

                        uint64_c previous_range=n;
                        //copying previous range from host to device.
                        if(hipMemcpy(device_previous_range,&previous_range,sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                        {
                                printf("ERROR: CANNOT COPY PRIME LIST FROM HOST TO DEVICE\n");
                                hipFree(device_input_list);
                                hipFree(device_prime_list);
                                hipFree(device_previous_range);
                                hipFree(device_max_prime_range);
                                hipFree(device_number_of_primes);
                                exit(0);
                        } 

                        printf("THE NUMBER OF PRIMES GENERATED: %llu \n",number_of_primes);
                        //to determine the maximum range a the calculated prime range can determine.
                        uint64_c max_prime_range = pow(n,2);

                        printf("MAXIMUM RANGE PRIMES BETWEEN 0 - %llu CAN DETERMINE IS %llu \n", n,max_prime_range);
                        
                        if(max_prime_range<=PRIME_RANGE){

                                if(hipMemcpy(device_max_prime_range,&max_prime_range,sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                                {
                                        printf("ERROR: CANNOT COPY MAX PRIME RANGE FROM HOST TO DEVICE\n");
                                        hipFree(device_input_list);
                                        hipFree(device_prime_list);
                                        hipFree(device_previous_range);
                                        hipFree(device_max_prime_range);
                                        hipFree(device_number_of_primes);
                                        exit(0);
                                } 

                                printf("CALCULATE PRIME NUMBERS BETWEEN %llu - %llu\n", previous_range,max_prime_range);
                                memsetting_range_of_input(input_list,max_prime_range);
                                calculatePrime(input_list, prime_list, previous_range, max_prime_range, number_of_primes);
                                number_of_primes = appending_prime(input_list, prime_list, previous_range, max_prime_range, number_of_primes);

                        }
                        else
                        {

                                if(hipMemcpy(device_max_prime_range,&prime_range,sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                                {
                                        printf("ERROR: CANNOT COPY MAX PRIME RANGE FROM HOST TO DEVICE\n");
                                        hipFree(device_input_list);
                                        hipFree(device_prime_list);
                                        hipFree(device_previous_range);
                                        hipFree(device_max_prime_range);
                                        hipFree(device_number_of_primes);
                                        exit(0);
                                } 
                                printf("CALCULATE PRIME NUMBERS BETWEEN %llu - %llu\n", previous_range,prime_range);
                                memsetting_range_of_input(input_list,prime_range);
                                calculatePrime(input_list, prime_list, previous_range, prime_range, number_of_primes);
                                number_of_primes = appending_prime(input_list, prime_list, previous_range, prime_range, number_of_primes);
                        }
                        printf("\n\n\n");
                        
                        //print_inputlist(input_list);
                        n=pow(n,2);
                       
                }


                printf("TOTAL NUMBER OF PRIMES GENERATED: %llu \n",number_of_primes);
                print_primelist(prime_list,number_of_primes);
//ending code for gpu.
        return 0;

 }



 uint64_c appending_prime(uint64_c* input_list, uint64_c* prime_list, uint64_c start_of_range,uint64_c end_of_range, uint64_c number_of_primes)
 {

                for(uint64_c i=start_of_range;i<end_of_range;i++)
                {

                        if(input_list[i]==0)
                        {
                                prime_list[number_of_primes] = i;
                                number_of_primes++;
                        }

                }

                return number_of_primes;

 }

void calculatePrime(uint64_c* input_list, uint64_c* prime_list, uint64_c start_of_range,uint64_c end_of_range, uint64_c number_of_primes)
{
        printf("--------CALCULATING PRIME NUMBERS from %llu to %llu --------\n", start_of_range,end_of_range);
       // print_primelist(prime_list,number_of_primes);
        for(uint64_c i=start_of_range;i<end_of_range;i++)
        {
                for(uint64_c j=0;j<number_of_primes;j++){


                
                        if(i % prime_list[j]==0)
                        {
                                input_list[i]=1;
                                
                               
                        }



                }
        }
        printf("-------- END CALCULATING PRIME NUMBERS--------\n");

}

void memsetting_range_of_input(uint64_c *input_list,uint64_c size)
{
        memset(input_list,0,size * sizeof(uint64_c));
}

void initializing_inputlist(uint64_c *input_list){

        for(int i=0;i<=INPUT_SIZE;i++)
        {
                input_list[i]=2;
        }


}

void print_inputlist(uint64_c *input_list)
{

        for(int i=0;i<INPUT_SIZE;i++)
        {
                printf("%d\t--->\t%llu\n", i,input_list[i]);
        }


}

void print_primelist(uint64_c *prime_list,uint64_c number_of_primes)
{

        for(int i=0;i<number_of_primes;i++)
        {
                printf("%llu\n",prime_list[i]);
        }


}


void copy_seed_primes(uint64_c *prime_list,int * seed_primelist,int num_of_seed)
{
        

        for(int i=0;i<num_of_seed;i++)
        {
                prime_list[i]=seed_primelist[i];
                
        }

}
int generate_seed_primes(int *input,int *primelist, uint64_c n)
{
    for (int p=2; p*p<=n; p++)
    {
         if (input[p] == 0)
        {
            for (int i=p*2; i<=n; i += p)
                input[i] = 1;
        }
    }




int i=0;
    for (int p=2; p<=n; p++){


       if (input[p]==0)
       {

          primelist[i]=p;
          i++;
       }



   }

   return i;

}


