#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include<hip/hip_runtime.h>

#define INPUT_SIZE 100000000
#define PRIME_RANGE 100000000
#define BLOCK_SIZE 32
typedef unsigned long long int uint64_c;


int generate_seed_primes(int*, int*, uint64_c);
void copy_seed_primes(uint64_c *,int *,int);
void print_primelist(uint64_c *, uint64_c);
void print_inputlist(uint64_c *);
void initializing_inputlist(uint64_c *);
void memsetting_range_of_input(uint64_c *,uint64_c);
void calculatePrime(uint64_c*, uint64_c*, uint64_c, uint64_c, uint64_c);
uint64_c appending_prime(uint64_c*, uint64_c*, uint64_c, uint64_c, uint64_c);


//KERNAL CODE GOES HERE!!


__global__ void prime_generator(uint64_c* device_input_list, uint64_c* device_prime_list, uint64_c* device_start_of_range,uint64_c* device_end_of_range, uint64_c* device_number_of_primes)
{
//	printf("------- INSIDE KERNEL ---------%llu-------%llu------\n",device_start_of_range[0],device_end_of_range[0]);

	int p= blockIdx.x * blockDim.x + threadIdx.x;
	int primeno= device_prime_list[p];
       for(uint64_c i=device_start_of_range[0];i<device_end_of_range[0];i++)
        {


                
                        if(i % primeno==0)
                        {
                                device_input_list[i]=1;
                                
                               
                        }



        }



}


//KERNAL CODE ENDS HERE!!!


int main()
{
	hipSetDevice(1);
// This code is just to generate the seed prime numbers
        int input_size=100;
        int *input;
        uint64_c n= 10 ;// seed prime list.
        int *seed_primelist;
        input=(int *)malloc(input_size*sizeof(int));
        seed_primelist=(int *)malloc(input_size*sizeof(int));
        int num_of_seed = generate_seed_primes(input,seed_primelist,n);


// seed prime list code ends here.


//Starting code for gpu.
        //declaring host variables.

        // declaring the ranges of the input size and the primes to be generated.

        uint64_c total_input_size = INPUT_SIZE;
        printf("TOTAL INPUT SIZE IS: %llu\n",total_input_size);
        uint64_c prime_range = PRIME_RANGE;
        printf("THE PRIMES WILL BE GENERATED FROM 0 - %llu\n",prime_range);

        printf("-------------------------------------------------------------------------\n\n\n");
        // creating the host array of input-list and primelist.
        uint64_c *input_list;
        uint64_c *prime_list;
        uint64_c number_of_primes= num_of_seed; //initializing the number of primes to the number of seed primes.
        input_list=(uint64_c *)malloc(total_input_size * sizeof(uint64_c));

        //setting all the values of the input list to -1.
        initializing_inputlist(input_list);

        prime_list=(uint64_c *)malloc(prime_range * sizeof(uint64_c));

        //copying the seed primes in prime_list.
                copy_seed_primes(prime_list,seed_primelist,num_of_seed);

         

        //creating the device array of input list and primelist
        uint64_c *device_input_list;
        uint64_c *device_prime_list;
        uint64_c *device_previous_range;
        uint64_c *device_max_prime_range;
        uint64_c *device_number_of_primes;


        //allocating memory in gpu.

        if(hipMalloc((void** )&device_input_list,total_input_size * sizeof(uint64_c))!=hipSuccess)
        {
                printf("ERROR: CANNOT ALLOCATE MEMORY IN GPU FOR INPUT LIST ------>> :) \n");
                exit(0);
        }

        if(hipMalloc((void** )&device_prime_list,prime_range * sizeof(uint64_c))!=hipSuccess)
        {
                printf("ERROR: CANNOT ALLOCATE MEMORY IN GPU FOR PRIME LIST\n");
                hipFree(device_input_list);
                exit(0);
        }

        if(hipMalloc((void** )&device_previous_range,sizeof(uint64_c))!=hipSuccess)
        {
                printf("ERROR: CANNOT ALLOCATE MEMORY IN GPU FOR PREVIOUS RANGE\n");
                hipFree(device_input_list);
                hipFree(device_prime_list);
                exit(0);
        }

        if(hipMalloc((void** )&device_max_prime_range,sizeof(uint64_c))!=hipSuccess)
        {
                printf("ERROR: CANNOT ALLOCATE MEMORY IN GPU FOR MAX PRIME RANGE\n");
                hipFree(device_input_list);
                hipFree(device_prime_list);
                hipFree(device_previous_range);
                exit(0);
        }

        if(hipMalloc((void** )&device_number_of_primes,sizeof(uint64_c))!=hipSuccess)
        {
                printf("ERROR: CANNOT ALLOCATE MEMORY IN GPU FOR NUMBER OF PRIMES\n");
                hipFree(device_input_list);
                hipFree(device_prime_list);
                hipFree(device_previous_range);
                hipFree(device_max_prime_range);
                exit(0);
        }

        //allocating memory in gpu completed.


        
        

                while(n<PRIME_RANGE){

                        //copying input list and prime list from host to device.

                        if(hipMemcpy(device_prime_list,prime_list,prime_range * sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                        {
                                printf("ERROR: CANNOT COPY PRIME LIST FROM HOST TO DEVICE\n");
                                hipFree(device_input_list);
                                hipFree(device_prime_list);
                                hipFree(device_previous_range);
                                hipFree(device_max_prime_range);
                                hipFree(device_number_of_primes);
                                exit(0);
                        } 

                        //copying input list and prime list from host to device completed.

                        //copying number of primes generated.
                        if(hipMemcpy(device_number_of_primes,&number_of_primes,sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                        {
                                printf("ERROR: CANNOT COPY NUMBER OF PRIMES FROM HOST TO DEVICE\n");
                                hipFree(device_input_list);
                                hipFree(device_prime_list);
                                hipFree(device_previous_range);
                                hipFree(device_max_prime_range);
                                hipFree(device_number_of_primes);
                                exit(0);
                        } 

                        uint64_c previous_range=n;
                        //copying previous range from host to device.
                        if(hipMemcpy(device_previous_range,&previous_range,sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                        {
                                printf("ERROR: CANNOT COPY PREVIOUS RANGE FROM HOST TO DEVICE\n");
                                hipFree(device_input_list);
                                hipFree(device_prime_list);
                                hipFree(device_previous_range);
                                hipFree(device_max_prime_range);
                                hipFree(device_number_of_primes);
                                exit(0);
                        } 

                        printf("THE NUMBER OF PRIMES GENERATED: %llu \n",number_of_primes);
                        //to determine the maximum range a the calculated prime range can determine.
                        uint64_c max_prime_range = pow(n,2);

                        printf("MAXIMUM RANGE PRIMES BETWEEN 0 - %llu CAN DETERMINE IS %llu \n", n,max_prime_range);
                        
                        if(max_prime_range<=PRIME_RANGE){

                                if(hipMemcpy(device_max_prime_range,&max_prime_range,sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                                {
                                        printf("ERROR: CANNOT COPY MAX PRIME RANGE FROM HOST TO DEVICE\n");
                                        hipFree(device_input_list);
                                        hipFree(device_prime_list);
                                        hipFree(device_previous_range);
                                        hipFree(device_max_prime_range);
                                        hipFree(device_number_of_primes);
                                        exit(0);
                                } 

                                printf("CALCULATE PRIME NUMBERS BETWEEN %llu - %llu\n", previous_range,max_prime_range);
                                memsetting_range_of_input(input_list,max_prime_range);
	                        if(hipMemcpy(device_input_list,input_list,total_input_size * sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
        	                {
                	                printf("ERROR: CANNOT COPY INPUT LIST FROM HOST TO DEVICE\n");
                        	        hipFree(device_input_list);
                             		hipFree(device_prime_list);
                                	hipFree(device_previous_range);
                                	hipFree(device_max_prime_range);
                                	hipFree(device_number_of_primes);
                                	exit(0);
                        	} 

                             //   calculatePrime(input_list, prime_list, previous_range, max_prime_range, number_of_primes);

                                prime_generator<<<BLOCK_SIZE,256>>>(device_input_list, device_prime_list, device_previous_range, device_max_prime_range, device_number_of_primes);
				hipError_t err = hipGetLastError();
				if (err != hipSuccess) 
   			        printf("Error: %s\n", hipGetErrorString(err));

				
                                //copying input list from device to host
                                if(hipMemcpy(input_list,device_input_list,total_input_size * sizeof(uint64_c),hipMemcpyDeviceToHost)!=hipSuccess)
                                {
                                        printf("ERROR: CANNOT COPY INPUT LIST FROM DEVICE TO HOST\n");
                                        hipFree(device_input_list);
                                        hipFree(device_prime_list);
                                        hipFree(device_previous_range);
                                        hipFree(device_max_prime_range);
                                        hipFree(device_number_of_primes);
                                        exit(0);
                                } 
				
                                number_of_primes = appending_prime(input_list, prime_list, previous_range, max_prime_range, number_of_primes);

                        }
                        else
                        {

                         	printf("IN ELSE PART.\n");  
			      if(hipMemcpy(device_max_prime_range,&prime_range,sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
                                {
                                        printf("ERROR: CANNOT COPY MAX PRIME RANGE FROM HOST TO DEVICE\n");
                                        hipFree(device_input_list);
                                        hipFree(device_prime_list);
                                        hipFree(device_previous_range);
                                        hipFree(device_max_prime_range);
                                        hipFree(device_number_of_primes);
                                        exit(0);
                                } 
                                printf("CALCULATE PRIME NUMBERS BETWEEN %llu - %llu\n", previous_range,prime_range);
                                memsetting_range_of_input(input_list,prime_range);

	                        if(hipMemcpy(device_input_list,input_list,total_input_size * sizeof(uint64_c),hipMemcpyHostToDevice)!=hipSuccess)
	                        {
        	                        printf("ERROR: CANNOT COPY INPUT LIST FROM HOST TO DEVICE\n");
                	                hipFree(device_input_list);
                        	        hipFree(device_prime_list);
                                	hipFree(device_previous_range);
                                	hipFree(device_max_prime_range);
                                	hipFree(device_number_of_primes);
                                	exit(0);
                        	} 
			//	printf("CHECKING TIME REQUIRED\n");

                            //    calculatePrime(input_list, prime_list, previous_range, prime_range, number_of_primes);
                                prime_generator<<<BLOCK_SIZE,256>>>(device_input_list, device_prime_list, device_previous_range, device_max_prime_range, device_number_of_primes);
				hipError_t err = hipGetLastError();
				if (err != hipSuccess) 
    				printf("Error: %s\n", hipGetErrorString(err));

                                //copying input list from device to host
                               if(hipMemcpy(input_list,device_input_list,total_input_size * sizeof(uint64_c),hipMemcpyDeviceToHost)!=hipSuccess)
                                {
                                        printf("ERROR: CANNOT COPY INPUT LIST FROM DEVICE TO HOST\n");
                                        hipFree(device_input_list);
                                        hipFree(device_prime_list);
                                        hipFree(device_previous_range);
                                        hipFree(device_max_prime_range);
                                        hipFree(device_number_of_primes);
                                        exit(0);
                                } 

                                number_of_primes = appending_prime(input_list, prime_list, previous_range, prime_range, number_of_primes);
                        }
                        printf("\n\n\n");
                        
                        //print_inputlist(input_list);
                        n=pow(n,2);
                       
                }


                printf("TOTAL NUMBER OF PRIMES GENERATED: %llu \n",number_of_primes);
                print_primelist(prime_list,number_of_primes);
//ending code for gpu.
        return 0;

 }



 uint64_c appending_prime(uint64_c* input_list, uint64_c* prime_list, uint64_c start_of_range,uint64_c end_of_range, uint64_c number_of_primes)
 {

                for(uint64_c i=start_of_range;i<end_of_range;i++)
                {

                        if(input_list[i]==0)
                        {
                                prime_list[number_of_primes] = i;
                                number_of_primes++;
                        }

                }

                return number_of_primes;

 }

void calculatePrime(uint64_c* input_list, uint64_c* prime_list, uint64_c start_of_range,uint64_c end_of_range, uint64_c number_of_primes)
{
        printf("--------CALCULATING PRIME NUMBERS from %llu to %llu --------\n", start_of_range,end_of_range);
       // print_primelist(prime_list,number_of_primes);
        for(uint64_c i=start_of_range;i<end_of_range;i++)
        {
                for(uint64_c j=0;j<number_of_primes;j++){


                
                        if(i % prime_list[j]==0)
                        {
                                input_list[i]=1;
                                
                               
                        }



                }
        }
        printf("-------- END CALCULATING PRIME NUMBERS--------\n");

}

void memsetting_range_of_input(uint64_c *input_list,uint64_c size)
{
        memset(input_list,0,size * sizeof(uint64_c));
}

void initializing_inputlist(uint64_c *input_list){

        for(int i=0;i<=INPUT_SIZE;i++)
        {
                input_list[i]=2;
        }


}

void print_inputlist(uint64_c *input_list)
{

        for(int i=0;i<INPUT_SIZE;i++)
        {
                printf("%d\t--->\t%llu\n", i,input_list[i]);
        }


}

void print_primelist(uint64_c *prime_list,uint64_c number_of_primes)
{

        for(int i=0;i<number_of_primes;i++)
        {
                printf("%llu\n",prime_list[i]);
        }


}


void copy_seed_primes(uint64_c *prime_list,int * seed_primelist,int num_of_seed)
{
        

        for(int i=0;i<num_of_seed;i++)
        {
                prime_list[i]=seed_primelist[i];
                
        }

}
int generate_seed_primes(int *input,int *primelist, uint64_c n)
{
    for (int p=2; p*p<=n; p++)
    {
         if (input[p] == 0)
        {
            for (int i=p*2; i<=n; i += p)
                input[i] = 1;
        }
    }




int i=0;
    for (int p=2; p<=n; p++){


       if (input[p]==0)
       {

          primelist[i]=p;
          i++;
       }



   }

   return i;

}

