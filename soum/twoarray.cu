#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>


__global__ void add(int *a, int *b,int * c)
{
	int col=10;	
	int i=0;
	int j=0;	

	for(i=0;i<10;i++){
		for(j=0;j<10;j++){

			*(c + i * col +j)= *(a + i + col + j) + *(b + i + col + j);			

		}
	}

}


int main()
{

	int row = 10;
	int col = 10;
	int *a;
	int *b;
	int *c;

	int *d_a;
	int *d_b;
	int *d_c;

	a=(int *) malloc(row * col * sizeof(int));
	b=(int *) malloc(row * col * sizeof(int));
	c=(int *) malloc(row * col * sizeof(int));

	int i,j;

	for(i=0;i<row;i++)
	{

		for(j=0;j<col;j++)
		{
			*(a + i * col + j)= 11;
			*(b + i * col + j)= 10;
		}

	}

	hipMalloc(&d_a,row*col*sizeof(int));
	hipMalloc(&d_b,row*col*sizeof(int));
	hipMalloc(&d_c, row*col*sizeof(int));	
	
	hipMemcpy(d_a,a,row*col*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,row*col*sizeof(int),hipMemcpyHostToDevice);
	
	add<<<1,100>>>(d_a,d_b,d_c);


	hipMemcpy(c,d_c,row*col*sizeof(int),hipMemcpyDeviceToHost);
	
	

	for(i=0;i<row;i++)
	{

		for(j=0;j<col;j++)
		{
			printf(" c[%d][%d] = %d\n",i,j, *(c + i * col + j) );
		}
		
	}

	return 0;
}
