
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#define INPUT_SIZE 100000000
#define PRIME_RANGE 10000000
typedef unsigned long long int uint64_c;


int generate_seed_primes(int*, int*, uint64_c);
void copy_seed_primes(uint64_c *,int *,int);
void print_primelist(uint64_c *, uint64_c);
void print_inputlist(uint64_c *);
void initializing_inputlist(uint64_c *);
void memsetting_range_of_input(uint64_c *,uint64_c);
void calculatePrime(uint64_c*, uint64_c*, uint64_c, uint64_c, uint64_c);
uint64_c appending_prime(uint64_c*, uint64_c*, uint64_c, uint64_c, uint64_c);


//KERNAL CODE GOES HERE!!





//KERNAL CODE ENDS HERE!!!


int main()
{

// This code is just to generate the seed prime numbers
        int input_size=100;
        int *input;
        uint64_c n= 10 ;// seed prime list.
        int *seed_primelist;
        input=(int *)malloc(input_size*sizeof(int));
        seed_primelist=(int *)malloc(input_size*sizeof(int));
        int num_of_seed = generate_seed_primes(input,seed_primelist,n);


// seed prime list code ends here.


//Starting code for gpu.
        //declaring host variables.

        // declaring the ranges of the input size and the primes to be generated.

        uint64_c total_input_size = INPUT_SIZE;
        printf("TOTAL INPUT SIZE IS: %llu\n",total_input_size);
        uint64_c prime_range = PRIME_RANGE;
        printf("THE PRIMES WILL BE GENERATED FROM 0 - %llu\n",prime_range);

        printf("-------------------------------------------------------------------------\n\n\n");
        // creating the host array of input-list and primelist.
        uint64_c *input_list;
        uint64_c *prime_list;
        uint64_c number_of_primes= num_of_seed; //initializing the number of primes to the number of seed primes.
        input_list=(uint64_c *)malloc(total_input_size * sizeof(uint64_c));

        //setting all the values of the input list to -1.
        initializing_inputlist(input_list);

        prime_list=(uint64_c *)malloc(prime_range * sizeof(uint64_c));

        //copying the seed primes in prime_list.
                copy_seed_primes(prime_list,seed_primelist,num_of_seed);

                
                while(n<PRIME_RANGE){

                        uint64_c previous_range=n;
                        printf("THE NUMBER OF PRIMES GENERATED: %llu \n",number_of_primes);
                        //to determine the maximum range a the calculated prime range can determine.
                        uint64_c max_prime_range = pow(n,2);
                        printf("MAXIMUM RANGE PRIMES BETWEEN 0 - %llu CAN DETERMINE IS %llu \n", n,max_prime_range);
                        
                        if(max_prime_range<=PRIME_RANGE){
                                printf("CALCULATE PRIME NUMBERS BETWEEN %llu - %llu\n", previous_range,max_prime_range);
                                memsetting_range_of_input(input_list,max_prime_range);
                                calculatePrime(input_list, prime_list, previous_range, max_prime_range, number_of_primes);
                                number_of_primes = appending_prime(input_list, prime_list, previous_range, max_prime_range, number_of_primes);

                        }
                        else
                        {
                                printf("CALCULATE PRIME NUMBERS BETWEEN %llu - %d\n", previous_range,PRIME_RANGE);
                                memsetting_range_of_input(input_list,PRIME_RANGE);
                                calculatePrime(input_list, prime_list, previous_range, PRIME_RANGE, number_of_primes);
                                number_of_primes = appending_prime(input_list, prime_list, previous_range, PRIME_RANGE, number_of_primes);
                        }
                        printf("\n\n\n");
                        
                        //print_inputlist(input_list);
                        n=pow(n,2);
                       
                }


                printf("TOTAL NUMBER OF PRIMES GENERATED: %llu \n",number_of_primes);
                print_primelist(prime_list,number_of_primes);
//ending code for gpu.
        return 0;

 }



 uint64_c appending_prime(uint64_c* input_list, uint64_c* prime_list, uint64_c start_of_range,uint64_c end_of_range, uint64_c number_of_primes)
 {

                for(uint64_c i=start_of_range;i<end_of_range;i++)
                {

                        if(input_list[i]==0)
                        {
                                prime_list[number_of_primes] = i;
                                number_of_primes++;
                        }

                }

                return number_of_primes;

 }

void calculatePrime(uint64_c* input_list, uint64_c* prime_list, uint64_c start_of_range,uint64_c end_of_range, uint64_c number_of_primes)
{
        printf("--------CALCULATING PRIME NUMBERS from %llu to %llu --------\n", start_of_range,end_of_range);
       // print_primelist(prime_list,number_of_primes);
        for(uint64_c i=start_of_range;i<end_of_range;i++)
        {
                for(uint64_c j=0;j<number_of_primes;j++){


                
                        if(i % prime_list[j]==0)
                        {
                                input_list[i]=1;
                                
                               
                        }



                }
        }
        printf("-------- END CALCULATING PRIME NUMBERS--------\n");

}

void memsetting_range_of_input(uint64_c *input_list,uint64_c size)
{
        memset(input_list,0,size * sizeof(uint64_c));
}

void initializing_inputlist(uint64_c *input_list){

        for(int i=0;i<=INPUT_SIZE;i++)
        {
                input_list[i]=2;
        }


}

void print_inputlist(uint64_c *input_list)
{

        for(int i=0;i<INPUT_SIZE;i++)
        {
                printf("%d\t--->\t%llu\n", i,input_list[i]);
        }


}

void print_primelist(uint64_c *prime_list,uint64_c number_of_primes)
{

        for(int i=0;i<number_of_primes;i++)
        {
                printf("%llu\n",prime_list[i]);
        }


}


void copy_seed_primes(uint64_c *prime_list,int * seed_primelist,int num_of_seed)
{
        

        for(int i=0;i<num_of_seed;i++)
        {
                prime_list[i]=seed_primelist[i];
                
        }

}
int generate_seed_primes(int *input,int *primelist, uint64_c n)
{
    for (int p=2; p*p<=n; p++)
    {
         if (input[p] == 0)
        {
            for (int i=p*2; i<=n; i += p)
                input[i] = 1;
        }
    }




int i=0;
    for (int p=2; p<=n; p++){


       if (input[p]==0)
       {

          primelist[i]=p;
          i++;
       }



   }

   return i;

}


